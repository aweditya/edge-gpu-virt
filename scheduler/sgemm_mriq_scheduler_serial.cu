#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <pthread.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <math.h>
#include <sys/time.h>
#include <malloc.h>
#include <vector>
#include <parboil.h>
#include "sgemm_kernel_sliced.cu"
#include "computeQ_sliced.cu"
#include "file.h"
#include "kernel.h"

#define DEVICE_RESET hipDeviceReset();

template <typename T>
void check(T result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        fprintf(stderr, "CUDA error at %s:%d code=%d \"%s\" \n",
                file, line, static_cast<unsigned int>(result), func);
        DEVICE_RESET
        // Make sure we call CUDA Device Reset before exiting
        exit(EXIT_FAILURE);
    }
}

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

// I/O routines
extern bool readColMajorMatrixFile(const char *fn, int &nr_row, int &nr_col, std::vector<float> &v);
extern bool writeColMajorMatrixFile(const char *fn, int, int, std::vector<float> &);

extern "C" void computeGold(float *, const float *, const float *, unsigned int, unsigned int, unsigned int);

void *launch_kernel_sgemm(void *thread_args)
{
    struct timeval t0, t1, dt;
    gettimeofday(&t0, NULL);

    float *dA, *dB, *dC;
    sgemm_args_t *args = (sgemm_args_t *)thread_args;

    // CUDA memory allocation
    std::vector<float> matC(args->matArow * args->matBcol);

    checkCudaErrors(hipMalloc((void **)&dA, args->A_sz));
    checkCudaErrors(hipMalloc((void **)&dB, args->B_sz));
    checkCudaErrors(hipMalloc((void **)&dC, args->C_sz));

    // Copy A and B^T into device memory
    checkCudaErrors(hipMemcpyAsync(dA, &(args->matA.front()), args->A_sz, hipMemcpyHostToDevice, args->stream));
    checkCudaErrors(hipMemcpyAsync(dB, &(args->matBT.front()), args->B_sz, hipMemcpyHostToDevice, args->stream));

    // Use standard sgemm interface
    regtileSgemm('N', 'T', args->matArow, args->matBcol, args->matAcol, 1.0f,
                 dA, args->matArow, dB, args->matBcol, 0.0f, dC, args->matArow, &(args->stream), &(args->kcb));

    checkCudaErrors(hipMemcpyAsync(&matC.front(), dC, args->C_sz, hipMemcpyDeviceToHost, args->stream));

    checkCudaErrors(hipFree(dA));
    checkCudaErrors(hipFree(dB));
    checkCudaErrors(hipFree(dC));

    gettimeofday(&t1, NULL);
    timersub(&t1, &t0, &dt);
    printf("(SGEMM, thread %ld) took %ld.%06ld sec\n", (long)pthread_self(), dt.tv_sec, dt.tv_usec);

    return NULL;
}

void *launch_kernel_mriq(void *thread_args)
{
    struct timeval t0, t1, dt;
    gettimeofday(&t0, NULL);

    mriq_args_t *args = (mriq_args_t *)thread_args;

    /* Create CPU data structures */
    createDataStructsCPU(args->numK, args->numX, &(args->phiMag), &(args->Qr), &(args->Qi));

    /* GPU section 1 (precompute PhiMag) */
    {
        /* Mirror several data structures on the device */
        float *phiR_d, *phiI_d;
        float *phiMag_d;

        checkCudaErrors(hipMalloc((void **)&phiR_d, args->numK * sizeof(float)));
        checkCudaErrors(hipMalloc((void **)&phiI_d, args->numK * sizeof(float)));

        checkCudaErrors(hipMemcpyAsync(phiR_d, args->phiR, args->numK * sizeof(float), hipMemcpyHostToDevice, args->stream));
        checkCudaErrors(hipMemcpyAsync(phiI_d, args->phiI, args->numK * sizeof(float), hipMemcpyHostToDevice, args->stream));

        checkCudaErrors(hipMalloc((void **)&phiMag_d, args->numK * sizeof(float)));

        hipDeviceSynchronize();

        computePhiMag_GPU(args->numK, phiR_d, phiI_d, phiMag_d, &(args->stream), &(args->kcb));

        hipDeviceSynchronize();

        checkCudaErrors(hipMemcpyAsync(args->phiMag, phiMag_d, args->numK * sizeof(float), hipMemcpyDeviceToHost, args->stream));

        checkCudaErrors(hipFree(phiMag_d));
        checkCudaErrors(hipFree(phiR_d));
        checkCudaErrors(hipFree(phiI_d));
    }

    // struct kValues *kVals;
    // kVals = (struct kValues *)calloc(args->numK, sizeof(struct kValues));
    // for (int k = 0; k < args->numK; k++)
    // {
    //     kVals[k].Kx = args->kx[k];
    //     kVals[k].Ky = args->ky[k];
    //     kVals[k].Kz = args->kz[k];
    //     kVals[k].PhiMag = args->phiMag[k];
    // }

    // /* GPU section 2 */
    // {
    //     float *x_d, *y_d, *z_d;
    //     float *Qr_d, *Qi_d;

    //     checkCudaErrors(hipMalloc((void **)&x_d, args->numX * sizeof(float)));
    //     checkCudaErrors(hipMemcpyAsync(x_d, args->x, args->numX * sizeof(float), hipMemcpyHostToDevice, args->stream));

    //     checkCudaErrors(hipMalloc((void **)&y_d, args->numX * sizeof(float)));
    //     checkCudaErrors(hipMemcpyAsync(y_d, args->y, args->numX * sizeof(float), hipMemcpyHostToDevice, args->stream));

    //     checkCudaErrors(hipMalloc((void **)&z_d, args->numX * sizeof(float)));
    //     checkCudaErrors(hipMemcpyAsync(z_d, args->z, args->numX * sizeof(float), hipMemcpyHostToDevice, args->stream));

    //     checkCudaErrors(hipMalloc((void **)&Qr_d, args->numX * sizeof(float)));
    //     checkCudaErrors(hipMemsetAsync((void *)Qr_d, 0, args->numX * sizeof(float), args->stream));

    //     checkCudaErrors(hipMalloc((void **)&Qi_d, args->numX * sizeof(float)));
    //     checkCudaErrors(hipMemsetAsync((void *)Qi_d, 0, args->numX * sizeof(float), args->stream));

    //     hipDeviceSynchronize();

    //     computeQ_GPU(args->numK, args->numX, x_d, y_d, z_d, kVals, Qr_d, Qi_d, &args->stream, args->kcb);

    //     hipDeviceSynchronize();

    //     checkCudaErrors(hipMemcpyAsync(args->Qr, Qr_d, args->numX * sizeof(float), hipMemcpyDeviceToHost, args->stream));
    //     checkCudaErrors(hipMemcpyAsync(args->Qi, Qi_d, args->numX * sizeof(float), hipMemcpyDeviceToHost, args->stream));

    //     checkCudaErrors(hipFree(x_d));
    //     checkCudaErrors(hipFree(y_d));
    //     checkCudaErrors(hipFree(z_d));
    //     checkCudaErrors(hipFree(Qr_d));
    //     checkCudaErrors(hipFree(Qi_d));
    // }

    // free(kVals);

    gettimeofday(&t1, NULL);
    timersub(&t1, &t0, &dt);
    printf("(MRI-Q, thread %ld) took %ld.%06ld sec\n", (long)pthread_self(), dt.tv_sec, dt.tv_usec);

    return NULL;
}

int main(int argc, char **argv)
{
    struct pb_Parameters *params;

    size_t A_sz, B_sz, C_sz;
    int matArow, matAcol;
    int matBrow, matBcol;
    std::vector<float> matA, matBT;

    /* Read command line. Expect 3 inputs: A, B and B^T
       in column-major layout*/
    params = pb_ReadParameters(&argc, argv);
    printf("%s %s %s\n", params->inpFiles[0], params->inpFiles[1], params->inpFiles[2]);
    if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] == NULL) || (params->inpFiles[2] == NULL) || (params->inpFiles[3] != NULL))
    {
        fprintf(stderr, "Expecting three input filenames\n");
        exit(-1);
    }

    /* Read in data */
    // load A
    readColMajorMatrixFile(params->inpFiles[1],
                           matArow, matAcol, matA);
    // copy A to device memory
    A_sz = matArow * matAcol * sizeof(float);

    // load B^T
    readColMajorMatrixFile(params->inpFiles[2],
                           matBcol, matBrow, matBT);

    B_sz = matBrow * matBcol * sizeof(float);

    // allocate space for C
    C_sz = matArow * matBcol * sizeof(float);

    float elapsed_time;

    const int num_threads = 2;
    pthread_t threads[num_threads];
    sgemm_args_t sgemm_args[num_threads];

    hipEvent_t start_event, stop_event;
    checkCudaErrors(hipEventCreate(&start_event));
    checkCudaErrors(hipEventCreate(&stop_event));

    for (int i = 0; i < num_threads; ++i)
    {
        kernel_control_block_init(&(sgemm_args[i].kcb));
        checkCudaErrors(hipStreamCreate(&(sgemm_args[i].stream)));
        sgemm_args[i].A_sz = A_sz;
        sgemm_args[i].B_sz = B_sz;
        sgemm_args[i].C_sz = C_sz;
        sgemm_args[i].matArow = matArow;
        sgemm_args[i].matAcol = matAcol;
        sgemm_args[i].matBrow = matBrow;
        sgemm_args[i].matBcol = matBcol;
        sgemm_args[i].matA = matA;
        sgemm_args[i].matBT = matBT;
    }

    hipEventRecord(start_event, 0);

    pthread_create(&threads[0], NULL, launch_kernel_sgemm, &sgemm_args[0]);
    while (true)
    {
	    pthread_mutex_lock(&(sgemm_args[0].kcb.kernel_lock));
	    sgemm_args[0].kcb.slicesToLaunch = 4;
	    sgemm_args[0].kcb.state = RUNNING;
	    pthread_mutex_unlock(&(sgemm_args[0].kcb.kernel_lock));
	    pthread_cond_signal(&(sgemm_args[0].kcb.kernel_signal));

	    if (sgemm_args[0].kcb.slicesLeft == 0)
		    break;
    }
    pthread_join(threads[0], NULL);

    pthread_create(&threads[1], NULL, launch_kernel_sgemm, &sgemm_args[1]);
    while (true)
    {
	    pthread_mutex_lock(&(sgemm_args[1].kcb.kernel_lock));
	    sgemm_args[1].kcb.slicesToLaunch = 1;
	    sgemm_args[1].kcb.state = RUNNING;
	    pthread_mutex_unlock(&(sgemm_args[1].kcb.kernel_lock));
	    pthread_cond_signal(&(sgemm_args[1].kcb.kernel_signal));

	    if (sgemm_args[1].kcb.slicesLeft == 0)
		    break;
    }
    pthread_join(threads[1], NULL);

    checkCudaErrors(hipEventRecord(stop_event, 0));
    checkCudaErrors(hipEventSynchronize(stop_event));
    checkCudaErrors(hipEventElapsedTime(&elapsed_time, start_event, stop_event));

    for (int i = 0; i < num_threads; ++i)
    {
        kernel_control_block_destroy(&(sgemm_args[i].kcb));
        hipStreamDestroy(sgemm_args[i].stream);
    }

    checkCudaErrors(hipEventDestroy(start_event));
    checkCudaErrors(hipEventDestroy(stop_event));

    printf("Measured time for sample = %.3fms\n", elapsed_time);
    DEVICE_RESET

    return 0;
}
