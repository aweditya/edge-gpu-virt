#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <malloc.h>
#include <vector>
#include <parboil.h>
#include "sgemm_kernel.cu"
// #include "sgemm_kernel_sliced.cu"

// I/O routines
extern bool readColMajorMatrixFile(const char *fn, int &nr_row, int &nr_col, std::vector<float> &v);
extern bool writeColMajorMatrixFile(const char *fn, int, int, std::vector<float> &);

extern "C" void computeGold(float *, const float *, const float *, unsigned int, unsigned int, unsigned int);

typedef struct thread_args
{
    hipStream_t *stream;
    size_t A_sz, B_sz, C_sz;
    int matArow, matAcol;
    int matBrow, matBcol;
    std::vector<float> matA, matBT;
} thread_args_t;

#define CHECK_ERROR(errorMessage)                                               \
    {                                                                           \
        hipError_t err = hipGetLastError();                                   \
        if (hipSuccess != err)                                                 \
        {                                                                       \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",   \
                    errorMessage, __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    }

void *launch_kernel(void *thread_args)
{
    float *dA, *dB, *dC;
    thread_args_t *args = (thread_args_t *)thread_args;

    // CUDA memory allocation
    std::vector<float> matC(args->matArow * args->matBcol);

    if (!(hipSuccess == hipMalloc((void **)&dA, args->A_sz)))
    {
        CHECK_ERROR("hipMalloc");
    }

    if (!(hipSuccess == hipMalloc((void **)&dB, args->B_sz)))
    {
        CHECK_ERROR("hipMalloc");
    }

    if (!(hipSuccess == hipMalloc((void **)&dC, args->C_sz)))
    {
        CHECK_ERROR("hipMalloc");
    }

    // Copy A and B^T into device memory
    if (!(hipSuccess == hipMemcpyAsync(dA, &(args->matA.front()), args->A_sz, hipMemcpyHostToDevice, *(args->stream))))
    {
        CHECK_ERROR("hipMemcpyAsync");
    }

    if (!(hipSuccess == hipMemcpyAsync(dB, &(args->matBT.front()), args->B_sz, hipMemcpyHostToDevice, *(args->stream))))
    {
        CHECK_ERROR("hipMemcpyAsync");
    }

    // Use standard sgemm interface
    regtileSgemm('N', 'T', args->matArow, args->matBcol, args->matAcol, 1.0f,
                 dA, args->matArow, dB, args->matBcol, 0.0f, dC, args->matArow, args->stream);

    if (!(hipSuccess == hipMemcpyAsync(&matC.front(), dC, args->C_sz, hipMemcpyDeviceToHost, *(args->stream))))
    {
        CHECK_ERROR("hipMemcpyAsync");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return NULL;
}

int main(int argc, char *argv[])
{
    struct pb_Parameters *params;

    size_t A_sz, B_sz, C_sz;
    int matArow, matAcol;
    int matBrow, matBcol;
    std::vector<float> matA, matBT;

    /* Read command line. Expect 3 inputs: A, B and B^T
       in column-major layout*/
    params = pb_ReadParameters(&argc, argv);
    printf("%s %s %s\n", params->inpFiles[0], params->inpFiles[1], params->inpFiles[2]);
    if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] == NULL) || (params->inpFiles[2] == NULL) || (params->inpFiles[3] != NULL))
    {
        fprintf(stderr, "Expecting three input filenames\n");
        exit(-1);
    }

    /* Read in data */
    // load A
    readColMajorMatrixFile(params->inpFiles[0],
                           matArow, matAcol, matA);
    // copy A to device memory
    A_sz = matArow * matAcol * sizeof(float);

    // load B^T
    readColMajorMatrixFile(params->inpFiles[2],
                           matBcol, matBrow, matBT);

    B_sz = matBrow * matBcol * sizeof(float);

    // allocate space for C
    C_sz = matArow * matBcol * sizeof(float);

    float elapsed_time;

    const int num_threads = 4;
    pthread_t threads[num_threads];
    thread_args_t args[num_threads];

    hipEvent_t start_event, stop_event;
    if (!(hipSuccess == hipEventCreate(&start_event)))
    {
        CHECK_ERROR("hipEventCreate");
    }

    if (!(hipSuccess == hipEventCreate(&stop_event)))
    {
        CHECK_ERROR("hipEventCreate");
    }

    for (int i = 0; i < num_threads; ++i)
    {
        if (!(hipSuccess == hipStreamCreate(args[i].stream)))
        {
            CHECK_ERROR("hipStreamCreate");
        }
        args[i].A_sz = A_sz;
        args[i].B_sz = B_sz;
        args[i].C_sz = C_sz;
        args[i].matArow = matArow;
        args[i].matAcol = matAcol;
        args[i].matBrow = matBrow;
        args[i].matBcol = matBcol;
        args[i].matA = matA;
        args[i].matBT = matBT;
    }

    hipEventRecord(start_event, 0);
    for (int i = 0; i < num_threads; ++i)
    {
        if (pthread_create(&threads[i], NULL, launch_kernel, &args[i]))
        {
            fprintf(stderr, "Error creating threadn");
            return 1;
        }
    }

    for (int i = 0; i < num_threads; ++i)
    {
        if (pthread_join(threads[i], NULL))
        {
            fprintf(stderr, "Error joining threadn");
            return 2;
        }
    }

    if (!(hipSuccess == hipEventRecord(stop_event, 0)))
    {
        CHECK_ERROR("hipEventRecord");
    }

    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);

    for (int i = 0; i < num_threads; ++i)
        hipStreamDestroy(*(args[i].stream));

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipDeviceReset();

    printf("Measured time for sample = %.3fms\n", elapsed_time);
    return 0;
}
