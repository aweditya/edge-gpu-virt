#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <math.h>

#define CHECK_ERROR(errorMessage)                                               \
    {                                                                           \
        hipError_t err = hipGetLastError();                                   \
        if (hipSuccess != err)                                                 \
        {                                                                       \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",   \
                    errorMessage, __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    }

__global__ void MatrixAdd(float *matA, float *matB, int width, dim3 blockOffset)
{
    int row = (blockIdx.x + blockOffset.x) * blockDim.x + threadIdx.x;
    int col = (blockIdx.y + blockOffset.y) * blockDim.y + threadIdx.y;

    matA[row * width + col] += matB[row * width + col];
}

float drand(float lo, float hi)
{
    return lo + (hi - lo) * rand() / RAND_MAX;
}

void *launch_kernel(void *thread_args)
{
    hipStream_t *stream = (hipStream_t *)thread_args;

    int width = 4;

    /* Grid dimension */
    dim3 gridConf(width, width);

    /* Block dimension */
    dim3 blockConf(width, width);

    int totalElements = gridConf.x * gridConf.y * blockConf.x * blockConf.y;
    printf("Matrices A and B of dimension (%d, %d) are being added\n", gridConf.x * blockConf.x, gridConf.y * blockConf.y);

    /* Allocate host memory for matA and matB */
    float *h_matA = (float *)malloc(totalElements * sizeof(float));
    float *h_matB = (float *)malloc(totalElements * sizeof(float));

    if (!(hipSuccess == hipHostMalloc((void **)&h_matA, totalElements * sizeof(float), hipHostMallocDefault))) // allocating memory on CPU
    {
        printf("cuda malloc host for h_matA failed!\n");
        CHECK_ERROR("hipHostMalloc");
    }

    if (!(hipSuccess == hipHostMalloc((void **)&h_matB, totalElements * sizeof(float), hipHostMallocDefault))) // allocating memory on CPU
    {
        printf("cuda malloc host for h_matB failed!\n");
        CHECK_ERROR("hipHostMalloc");
    }

    /* Initialize matA and matB randomly */
    for (int i = 0; i < totalElements; ++i)
    {
        h_matA[i] = drand(0.0, 1.0);
        h_matB[i] = drand(0.0, 1.0);
    }

    /* Allocate device memory for matA and matB */
    float *d_matA, *d_matB;

    if (!(hipSuccess == hipMalloc((void **)&d_matA, totalElements * sizeof(float)))) // allocating memory on GPU
    {
        printf("cuda malloc for d_matA failed!\n");
        CHECK_ERROR("hipMalloc");
    }
    if (!(hipSuccess == hipMalloc((void **)&d_matB, totalElements * sizeof(float)))) // allocating memory on GPU
    {
        printf("cuda malloc for d_matB failed!\n");
        CHECK_ERROR("hipMalloc");
    }

    /* copy data from host to device */
    if (!(hipSuccess == hipMemcpyAsync(d_matA, h_matA, totalElements * sizeof(float), hipMemcpyHostToDevice, *stream)))
    {
        printf("hipMemcpyAsync for (d_matA, h_matA) failed!\n");
        CHECK_ERROR("hipMemcpyAsync");
    }

    if (!(hipSuccess == hipMemcpyAsync(d_matB, h_matB, totalElements * sizeof(float), hipMemcpyHostToDevice, *stream)))
    {
        printf("hipMemcpyAsync for (d_matB, h_matB) failed!\n");
        CHECK_ERROR("hipMemcpyAsync");
    }

    /* Sliced grid dimension: 8x1 */
    dim3 sGridConf(width / 4, width / 2);
    dim3 blockOffset(0, 0);
    while (blockOffset.x < gridConf.x && blockOffset.y < gridConf.y)
    {
        // printf("Calling slice with blockOffset (%d, %d)\n", blockOffset.x, blockOffset.y);
        MatrixAdd<<<sGridConf, blockConf, 0, *stream>>>(d_matA, d_matB, width * width, blockOffset);
        blockOffset.x += sGridConf.x;
        while (blockOffset.x >= gridConf.x)
        {
            blockOffset.x -= gridConf.x;
            blockOffset.y += sGridConf.y;
        }
    }

    /* copy result from device to host */
    if (!(hipSuccess == hipMemcpyAsync(h_matA, d_matA, totalElements * sizeof(float), hipMemcpyDeviceToHost, *stream)))
    {
        printf("hipMemcpy for (h_matA, d_matA) failed!\n");
        CHECK_ERROR("hipMemcpy");
    }

    hipHostFree(h_matA);
    hipHostFree(h_matB);
    hipFree(d_matA);
    hipFree(d_matB);

    return NULL;
}

int main(int argc, char *argv[])
{
    srand(0);

    const int num_threads = 4;
    pthread_t threads[num_threads];
    hipStream_t streams[num_threads];

    for (int i = 0; i < num_threads; ++i)
        hipStreamCreate(&streams[i]);

    for (int i = 0; i < num_threads; ++i)
    {
        if (pthread_create(&threads[i], NULL, launch_kernel, &streams[i]))
        {
            fprintf(stderr, "Error creating threadn");
            return 1;
        }
    }

    for (int i = 0; i < num_threads; ++i)
    {
        if (pthread_join(threads[i], NULL))
        {
            fprintf(stderr, "Error joining threadn");
            return 2;
        }
    }

    for (int i = 0; i < num_threads; ++i)
        hipStreamDestroy(streams[i]);

    hipDeviceReset();

    return 0;
}
