#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <math.h>

#define CHECK_ERROR(errorMessage)                                               \
    {                                                                           \
        hipError_t err = hipGetLastError();                                   \
        if (hipSuccess != err)                                                 \
        {                                                                       \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",   \
                    errorMessage, __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    }

__global__ void MatrixAdd(float *matA, float *matB, int width, dim3 blockOffset)
{
    int row = (blockIdx.x + blockOffset.x) * blockDim.x + threadIdx.x;
    int col = (blockIdx.y + blockOffset.y) * blockDim.y + threadIdx.y;

    matA[row * width + col] += matB[row * width + col];
}

float drand(float lo, float hi)
{
    return lo + (hi - lo) * rand() / RAND_MAX;
}

void *launch_kernel(void *thread_args)
{
    hipStream_t *stream = (hipStream_t *)thread_args;

    int width = 16;

    /* Grid dimension */
    dim3 gridConf(width, width);

    /* Block dimension */
    dim3 blockConf(width, width);

    int totalElements = gridConf.x * gridConf.y * blockConf.x * blockConf.y;
    printf("Matrices A and B of dimension (%d, %d) are being added\n", gridConf.x * blockConf.x, gridConf.y * blockConf.y);

    /* Allocate host memory for matA and matB */
    float *h_matA, *h_matB;

    if (!(hipSuccess == hipHostMalloc((void **)&h_matA, totalElements * sizeof(float), hipHostMallocDefault))) // allocating memory on CPU
    {
        CHECK_ERROR("hipHostMalloc");
    }

    if (!(hipSuccess == hipHostMalloc((void **)&h_matB, totalElements * sizeof(float), hipHostMallocDefault))) // allocating memory on CPU
    {
        CHECK_ERROR("hipHostMalloc");
    }

    /* Initialize matA and matB randomly */
    for (int i = 0; i < totalElements; ++i)
    {
        h_matA[i] = drand(0.0, 1.0);
        h_matB[i] = drand(0.0, 1.0);
    }

    /* Allocate device memory for matA and matB */
    float *d_matA, *d_matB;

    if (!(hipSuccess == hipMalloc((void **)&d_matA, totalElements * sizeof(float)))) // allocating memory on GPU
    {
        CHECK_ERROR("hipMalloc");
    }
    if (!(hipSuccess == hipMalloc((void **)&d_matB, totalElements * sizeof(float)))) // allocating memory on GPU
    {
        CHECK_ERROR("hipMalloc");
    }

    /* copy data from host to device */
    if (!(hipSuccess == hipMemcpyAsync(d_matA, h_matA, totalElements * sizeof(float), hipMemcpyHostToDevice, *stream)))
    {
        CHECK_ERROR("hipMemcpyAsync");
    }

    if (!(hipSuccess == hipMemcpyAsync(d_matB, h_matB, totalElements * sizeof(float), hipMemcpyHostToDevice, *stream)))
    {
        CHECK_ERROR("hipMemcpyAsync");
    }

    /* Sliced grid dimension: 8x1 */
    dim3 sGridConf(width, width);
    dim3 blockOffset(0, 0);
    while (blockOffset.x < gridConf.x && blockOffset.y < gridConf.y)
    {
        // printf("Calling slice with blockOffset (%d, %d)\n", blockOffset.x, blockOffset.y);
        MatrixAdd<<<sGridConf, blockConf, 0, *stream>>>(d_matA, d_matB, width * width, blockOffset);
        blockOffset.x += sGridConf.x;
        while (blockOffset.x >= gridConf.x)
        {
            blockOffset.x -= gridConf.x;
            blockOffset.y += sGridConf.y;
        }
    }

    /* copy result from device to host */
    if (!(hipSuccess == hipMemcpyAsync(h_matA, d_matA, totalElements * sizeof(float), hipMemcpyDeviceToHost, *stream)))
    {
        CHECK_ERROR("hipMemcpy");
    }

    hipHostFree(h_matA);
    hipHostFree(h_matB);
    hipFree(d_matA);
    hipFree(d_matB);

    return NULL;
}

int main(int argc, char *argv[])
{
    srand(0);
    float elapsed_time;

    const int num_threads = 8;
    pthread_t threads[num_threads];
    hipStream_t streams[num_threads];

    hipEvent_t start_event, stop_event;
    if (!(hipSuccess == hipEventCreate(&start_event)))
    {
        CHECK_ERROR("hipEventCreate");
    }

    if (!(hipSuccess == hipEventCreate(&stop_event)))
    {
        CHECK_ERROR("hipEventCreate");
    }

    for (int i = 0; i < num_threads; ++i)
        hipStreamCreate(&streams[i]);

    hipEventRecord(start_event, 0);
    for (int i = 0; i < num_threads; ++i)
    {
        if (pthread_create(&threads[i], NULL, launch_kernel, &streams[i]))
        {
            fprintf(stderr, "Error creating threadn");
            return 1;
        }
    }

    for (int i = 0; i < num_threads; ++i)
    {
        if (pthread_join(threads[i], NULL))
        {
            fprintf(stderr, "Error joining threadn");
            return 2;
        }
    }

    if (!(hipSuccess == hipEventRecord(stop_event, 0)))
    {
        CHECK_ERROR("hipEventRecord");
    }

    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);

    for (int i = 0; i < num_threads; ++i)
        hipStreamDestroy(streams[i]);

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipDeviceReset();

    printf("Measured time for sample = %.3fms\n", elapsed_time);
    return 0;
}
