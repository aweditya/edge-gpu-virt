#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <math.h>

#define CHECK_ERROR(errorMessage)                                               \
    {                                                                           \
        hipError_t err = hipGetLastError();                                   \
        if (hipSuccess != err)                                                 \
        {                                                                       \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",   \
                    errorMessage, __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    }

__global__ void MatrixAdd(float *matA, float *matB, int width, dim3 blockOffset)
{
    int row = (blockIdx.x + blockOffset.x) * blockDim.x + threadIdx.x;
    int col = (blockIdx.y + blockOffset.y) * blockDim.y + threadIdx.y;

    matA[row * width + col] += matB[row * width + col];
}

float drand(float lo, float hi)
{
    return lo + (hi - lo) * rand() / RAND_MAX;
}

void *launch_kernel(void *thread_args)
{
    int width = 16;

    /* Grid dimension */
    dim3 gridConf(width, width);

    /* Block dimension */
    dim3 blockConf(width, width);

    /* Sliced grid dimension: 8x1 */
    dim3 sGridConf(width / 4, width / 2);

    int totalElements = gridConf.x * gridConf.y * blockConf.x * blockConf.y;
    int numKernels = (gridConf.x * gridConf.y) / (sGridConf.x * sGridConf.y);
    printf("Matrices A and B of dimension (%d, %d) are being added with %d kernels\n", gridConf.x * blockConf.x, gridConf.y * blockConf.y, numKernels);

    // Create a new stream for each kernel slice
    hipStream_t *streams = (hipStream_t *)malloc(numKernels * sizeof(hipStream_t));

    for (int i = 0; i < numKernels; ++i)
    {
        if (!(hipSuccess == hipStreamCreate(&(streams[i]))))
        {
            CHECK_ERROR("hipStreamCreate");
        }
    }

    /* Allocate host memory for matA and matB */
    float *h_matA, *h_matB;

    if (!(hipSuccess == hipHostMalloc((void **)&h_matA, totalElements * sizeof(float), hipHostMallocDefault))) // allocating memory on CPU
    {
        CHECK_ERROR("hipHostMalloc");
    }

    if (!(hipSuccess == hipHostMalloc((void **)&h_matB, totalElements * sizeof(float), hipHostMallocDefault))) // allocating memory on CPU
    {
        CHECK_ERROR("hipHostMalloc");
    }

    /* Initialize matA and matB randomly */
    for (int i = 0; i < totalElements; ++i)
    {
        h_matA[i] = drand(0.0, 1.0);
        h_matB[i] = drand(0.0, 1.0);
    }

    /* Allocate device memory for matA and matB */
    float *d_matA, *d_matB;

    if (!(hipSuccess == hipMalloc((void **)&d_matA, totalElements * sizeof(float)))) // allocating memory on GPU
    {
        CHECK_ERROR("hipMalloc");
    }
    if (!(hipSuccess == hipMalloc((void **)&d_matB, totalElements * sizeof(float)))) // allocating memory on GPU
    {
        CHECK_ERROR("hipMalloc");
    }

    int currentKernel = 0; // Assigning each kernel slice to a unique stream
    int workPerKernel = totalElements / numKernels;
    dim3 blockOffset(0, 0);
    while (blockOffset.x < gridConf.x && blockOffset.y < gridConf.y)
    {
        // printf("Calling slice with blockOffset (%d, %d)\n", blockOffset.x, blockOffset.y);

        /* copy data from host to device */
        if (!(hipSuccess == hipMemcpyAsync(d_matA + currentKernel * workPerKernel, h_matA + currentKernel * workPerKernel, workPerKernel * sizeof(float), hipMemcpyHostToDevice, streams[currentKernel])))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        if (!(hipSuccess == hipMemcpyAsync(d_matB + currentKernel * workPerKernel, h_matB + currentKernel * workPerKernel, workPerKernel * sizeof(float), hipMemcpyHostToDevice, streams[currentKernel])))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        MatrixAdd<<<sGridConf, blockConf, 0, streams[currentKernel]>>>(d_matA, d_matB, width * width, blockOffset);
        blockOffset.x += sGridConf.x;
        while (blockOffset.x >= gridConf.x)
        {
            blockOffset.x -= gridConf.x;
            blockOffset.y += sGridConf.y;
        }

        /* copy result from device to host */
        if (!(hipSuccess == hipMemcpyAsync(h_matA + currentKernel * workPerKernel, d_matA + currentKernel * workPerKernel, workPerKernel * sizeof(float), hipMemcpyDeviceToHost, streams[currentKernel])))
        {
            CHECK_ERROR("hipMemcpy");
        }

        ++currentKernel;
    }

    for (int i = 0; i < numKernels; ++i)
    {
        if (!(hipSuccess == hipStreamDestroy(streams[i])))
        {
            CHECK_ERROR("hipStreamDestroy");
        }
    }

    free(streams);
    hipHostFree(h_matA);
    hipHostFree(h_matB);
    hipFree(d_matA);
    hipFree(d_matB);

    return NULL;
}

int main(int argc, char *argv[])
{
    srand(0);
    float elapsed_time;

    const int num_threads = 8;
    pthread_t threads[num_threads];

    hipEvent_t start_event, stop_event;
    if (!(hipSuccess == hipEventCreate(&start_event)))
    {
        CHECK_ERROR("hipEventCreate");
    }

    if (!(hipSuccess == hipEventCreate(&stop_event)))
    {
        CHECK_ERROR("hipEventCreate");
    }

    hipEventRecord(start_event, 0);
    for (int i = 0; i < num_threads; ++i)
    {
        if (pthread_create(&threads[i], NULL, launch_kernel, NULL))
        {
            fprintf(stderr, "Error creating threadn");
            return 1;
        }
    }

    for (int i = 0; i < num_threads; ++i)
    {
        if (pthread_join(threads[i], NULL))
        {
            fprintf(stderr, "Error joining threadn");
            return 2;
        }
    }

    if (!(hipSuccess == hipEventRecord(stop_event, 0)))
    {
        CHECK_ERROR("hipEventRecord");
    }

    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipDeviceReset();

    printf("Measured time for sample = %.3fms\n", elapsed_time);
    return 0;
}
