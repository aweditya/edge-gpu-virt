#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <math.h>

#define CHECK_ERROR(errorMessage)                                               \
    {                                                                           \
        hipError_t err = hipGetLastError();                                   \
        if (hipSuccess != err)                                                 \
        {                                                                       \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",   \
                    errorMessage, __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    }

__global__ void MatrixAdd(float *matA, float *matB, int width, dim3 blockOffset)
{
    int row = (blockIdx.x + blockOffset.x) * blockDim.x + threadIdx.x;
    int col = (blockIdx.y + blockOffset.y) * blockDim.y + threadIdx.y;

    matA[row * width + col] += matB[row * width + col];
}

float drand(float lo, float hi)
{
    return lo + (hi - lo) * rand() / RAND_MAX;
}

void *launch_kernel(void *thread_args)
{
    int width = 4;

    /* Grid dimension */
    dim3 gridConf(width, width);

    /* Block dimension */
    dim3 blockConf(width, width);

    int totalElements = gridConf.x * gridConf.y * blockConf.x * blockConf.y;
    printf("Matrices A and B of dimension (%d, %d) are being added\n", gridConf.x * blockConf.x, gridConf.y * blockConf.y);

    /* Allocate host memory for matA and matB */
    float *h_matA = (float *)malloc(totalElements * sizeof(float));
    float *h_matB = (float *)malloc(totalElements * sizeof(float));

    /* Initialize matA and matB randomly */
    for (int i = 0; i < totalElements; ++i)
    {
        h_matA[i] = drand(0.0, 1.0);
        h_matB[i] = drand(0.0, 1.0);
    }

    /* Allocate device memory for matA and matB */
    float *d_matA, *d_matB;

    if (!(hipSuccess == hipMalloc((void **)&d_matA, totalElements * sizeof(float)))) // allocating memory on GPU
    {
        printf("cuda malloc for d_matA failed!\n");
        CHECK_ERROR("hipMalloc");
    }
    if (!(hipSuccess == hipMalloc((void **)&d_matB, totalElements * sizeof(float)))) // allocating memory on GPU
    {
        printf("cuda malloc for d_matB failed!\n");
        CHECK_ERROR("hipMalloc");
    }

    /* copy data from host to device */
    if (!(hipSuccess == hipMemcpy(d_matA, h_matA, totalElements * sizeof(float), hipMemcpyHostToDevice)))
    {
        printf("hipMemcpyAsync for (d_matA, h_matA) failed!\n");
        CHECK_ERROR("hipMemcpyAsync");
    }

    if (!(hipSuccess == hipMemcpy(d_matB, h_matB, totalElements * sizeof(float), hipMemcpyHostToDevice)))
    {
        printf("hipMemcpyAsync for (d_matB, h_matB) failed!\n");
        CHECK_ERROR("hipMemcpyAsync");
    }

    /* Sliced grid dimension: 8x1 */
    dim3 sGridConf(width / 4, width / 2);
    dim3 blockOffset(0, 0);
    while (blockOffset.x < gridConf.x && blockOffset.y < gridConf.y)
    {
        // printf("Calling slice with blockOffset (%d, %d)\n", blockOffset.x, blockOffset.y);
        MatrixAdd<<<sGridConf, blockConf>>>(d_matA, d_matB, width * width, blockOffset);
        blockOffset.x += sGridConf.x;
        while (blockOffset.x >= gridConf.x)
        {
            blockOffset.x -= gridConf.x;
            blockOffset.y += sGridConf.y;
        }
    }

    /* copy result from device to host */
    if (!(hipSuccess == hipMemcpy(h_matA, d_matA, totalElements * sizeof(float), hipMemcpyDeviceToHost)))
    {
        printf("hipMemcpy for (h_matA, d_matA) failed!\n");
        CHECK_ERROR("hipMemcpy");
    }

    free(h_matA);
    free(h_matB);
    hipFree(d_matA);
    hipFree(d_matB);

    return NULL;
}

int main(int argc, char *argv[])
{
    srand(0);

    const int num_threads = 8;
    for (int i = 0; i < num_threads; ++i)
    {
        launch_kernel(NULL);
    }

    hipDeviceReset();

    return 0;
}
