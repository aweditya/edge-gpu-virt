#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <malloc.h>
#include <vector>
#include <parboil.h>
#include "sgemm_kernel_sliced.cu"
#include "computeQ_sliced.cu"
#include "file.h"

// I/O routines
extern bool readColMajorMatrixFile(const char *fn, int &nr_row, int &nr_col, std::vector<float> &v);
extern bool writeColMajorMatrixFile(const char *fn, int, int, std::vector<float> &);

extern "C" void computeGold(float *, const float *, const float *, unsigned int, unsigned int, unsigned int);

typedef struct thread_args_sgemm
{
    hipStream_t *stream;
    size_t A_sz, B_sz, C_sz;
    int matArow, matAcol;
    int matBrow, matBcol;
    std::vector<float> matA, matBT;
} thread_args_sgemm_t;

typedef struct thread_args_mriq
{
    hipStream_t *stream;
    int numX, numK;      /* Number of X and K values */
    float *kx, *ky, *kz; /* K trajectory (3D vectors) */
    float *x, *y, *z;    /* X coordinates (3D vectors) */
    float *phiR, *phiI;  /* Phi values (complex) */
    float *phiMag;       /* Magnitude of Phi */
    float *Qr, *Qi;      /* Q signal (complex) */

} thread_args_mriq_t;

#define CHECK_ERROR(errorMessage)                                               \
    {                                                                           \
        hipError_t err = hipGetLastError();                                   \
        if (hipSuccess != err)                                                 \
        {                                                                       \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",   \
                    errorMessage, __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    }

void *launch_kernel_sgemm(void *thread_args)
{
    float *dA, *dB, *dC;
    thread_args_sgemm_t *args = (thread_args_sgemm_t *)thread_args;

    // CUDA memory allocation
    std::vector<float> matC(args->matArow * args->matBcol);

    if (!(hipSuccess == hipMalloc((void **)&dA, args->A_sz)))
    {
        CHECK_ERROR("hipMalloc");
    }

    if (!(hipSuccess == hipMalloc((void **)&dB, args->B_sz)))
    {
        CHECK_ERROR("hipMalloc");
    }

    if (!(hipSuccess == hipMalloc((void **)&dC, args->C_sz)))
    {
        CHECK_ERROR("hipMalloc");
    }

    // Copy A and B^T into device memory
    if (!(hipSuccess == hipMemcpyAsync(dA, &(args->matA.front()), args->A_sz, hipMemcpyHostToDevice, *(args->stream))))
    {
        CHECK_ERROR("hipMemcpyAsync");
    }

    if (!(hipSuccess == hipMemcpyAsync(dB, &(args->matBT.front()), args->B_sz, hipMemcpyHostToDevice, *(args->stream))))
    {
        CHECK_ERROR("hipMemcpyAsync");
    }

    // Use standard sgemm interface
    regtileSgemm('N', 'T', args->matArow, args->matBcol, args->matAcol, 1.0f,
                 dA, args->matArow, dB, args->matBcol, 0.0f, dC, args->matArow, args->stream);

    if (!(hipSuccess == hipMemcpyAsync(&matC.front(), dC, args->C_sz, hipMemcpyDeviceToHost, *(args->stream))))
    {
        CHECK_ERROR("hipMemcpyAsync");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return NULL;
}

void *launch_kernel_mriq(void *thread_args)
{
    thread_args_mriq_t *args = (thread_args_mriq_t *)thread_args;

    /* Create CPU data structures */
    createDataStructsCPU(args->numK, args->numX, &(args->phiMag), &(args->Qr), &(args->Qi));

    /* GPU section 1 (precompute PhiMag) */
    {
        /* Mirror several data structures on the device */
        float *phiR_d, *phiI_d;
        float *phiMag_d;

        if (!(hipSuccess == hipMalloc((void **)&phiR_d, args->numK * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        if (!(hipSuccess == hipMalloc((void **)&phiI_d, args->numK * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        if (!(hipSuccess == hipMemcpyAsync(phiR_d, args->phiR, args->numK * sizeof(float), hipMemcpyHostToDevice, *(args->stream))))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        if (!(hipSuccess == hipMemcpyAsync(phiI_d, args->phiI, args->numK * sizeof(float), hipMemcpyHostToDevice, *(args->stream))))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        if (!(hipSuccess == hipMalloc((void **)&phiMag_d, args->numK * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        hipDeviceSynchronize();

        computePhiMag_GPU(args->numK, phiR_d, phiI_d, phiMag_d, args->stream);

        hipDeviceSynchronize();

        if (!(hipSuccess == hipMemcpyAsync(args->phiMag, phiMag_d, args->numK * sizeof(float), hipMemcpyDeviceToHost, *(args->stream))))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        hipFree(phiMag_d);
        hipFree(phiR_d);
        hipFree(phiI_d);
    }

    struct kValues *kVals;
    kVals = (struct kValues *)calloc(args->numK, sizeof(struct kValues));
    for (int k = 0; k < args->numK; k++)
    {
        kVals[k].Kx = args->kx[k];
        kVals[k].Ky = args->ky[k];
        kVals[k].Kz = args->kz[k];
        kVals[k].PhiMag = args->phiMag[k];
    }

    /* GPU section 2 */
    {
        float *x_d, *y_d, *z_d;
        float *Qr_d, *Qi_d;

        if (!(hipSuccess == hipMalloc((void **)&x_d, args->numX * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        if (!(hipSuccess == hipMemcpyAsync(x_d, args->x, args->numX * sizeof(float), hipMemcpyHostToDevice, *(args->stream))))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        if (!(hipSuccess == hipMalloc((void **)&y_d, args->numX * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        if (!(hipSuccess == hipMemcpyAsync(y_d, args->y, args->numX * sizeof(float), hipMemcpyHostToDevice, *(args->stream))))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        if (!(hipSuccess == hipMalloc((void **)&z_d, args->numX * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        if (!(hipSuccess == hipMemcpyAsync(z_d, args->z, args->numX * sizeof(float), hipMemcpyHostToDevice, *(args->stream))))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        if (!(hipSuccess == hipMalloc((void **)&Qr_d, args->numX * sizeof(float))))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        if (!(hipSuccess == hipMemsetAsync((void *)Qr_d, 0, args->numX * sizeof(float), *(args->stream))))
        {
            CHECK_ERROR("hipMemsetAsync");
        }

        if (!(hipSuccess == hipMalloc((void **)&Qi_d, args->numX * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        if (!(hipSuccess == hipMemsetAsync((void *)Qi_d, 0, args->numX * sizeof(float), *(args->stream))))
        {
            CHECK_ERROR("hipMemsetAsync");
        }

        hipDeviceSynchronize();

        computeQ_GPU(args->numK, args->numX, x_d, y_d, z_d, kVals, Qr_d, Qi_d, args->stream);

        hipDeviceSynchronize();

        if (!(hipSuccess == hipMemcpyAsync(args->Qr, Qr_d, args->numX * sizeof(float), hipMemcpyDeviceToHost, *(args->stream))))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        if (!(hipSuccess == hipMemcpyAsync(args->Qi, Qi_d, args->numX * sizeof(float), hipMemcpyDeviceToHost, *(args->stream))))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        hipFree(x_d);
        hipFree(y_d);
        hipFree(z_d);
        hipFree(Qr_d);
        hipFree(Qi_d);
    }

    free(kVals);
    return NULL;
}

int main(int argc, char *argv[])
{
    struct pb_Parameters *params;

    size_t A_sz, B_sz, C_sz;
    int matArow, matAcol;
    int matBrow, matBcol;
    std::vector<float> matA, matBT;

    int numX, numK;      /* Number of X and K values */
    int original_numK;   /* Number of K values in input file */
    float *kx, *ky, *kz; /* K trajectory (3D vectors) */
    float *x, *y, *z;    /* X coordinates (3D vectors) */
    float *phiR, *phiI;  /* Phi values (complex) */
    float *phiMag;       /* Magnitude of Phi */
    float *Qr, *Qi;      /* Q signal (complex) */

    /* Read command line. Expect 3 inputs: A, B and B^T
       in column-major layout*/
    params = pb_ReadParameters(&argc, argv);
    printf("%s %s %s\n", params->inpFiles[0], params->inpFiles[1], params->inpFiles[2]);
    if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] == NULL) || (params->inpFiles[2] == NULL) || (params->inpFiles[3] != NULL))
    {
        fprintf(stderr, "Expecting three input filenames\n");
        exit(-1);
    }

    inputData(params->inpFiles[0],
              &original_numK, &numX,
              &kx, &ky, &kz,
              &x, &y, &z,
              &phiR, &phiI);

    numK = original_numK;

    printf("%d pixels in output; %d samples in trajectory; using %d samples\n",
           numX, original_numK, numK);

    /* Read in data */
    // load A
    readColMajorMatrixFile(params->inpFiles[1],
                           matArow, matAcol, matA);
    // copy A to device memory
    A_sz = matArow * matAcol * sizeof(float);

    // load B^T
    readColMajorMatrixFile(params->inpFiles[2],
                           matBcol, matBrow, matBT);

    B_sz = matBrow * matBcol * sizeof(float);

    // allocate space for C
    C_sz = matArow * matBcol * sizeof(float);

    float elapsed_time;

    const int num_threads = 4;
    pthread_t threads[num_threads];
    thread_args_sgemm_t sgemm_args[num_threads / 2];
    thread_args_mriq_t mriq_args[num_threads / 2];

    hipEvent_t start_event, stop_event;
    if (!(hipSuccess == hipEventCreate(&start_event)))
    {
        CHECK_ERROR("hipEventCreate");
    }

    if (!(hipSuccess == hipEventCreate(&stop_event)))
    {
        CHECK_ERROR("hipEventCreate");
    }

    for (int i = 0; i < num_threads / 2; ++i)
    {
        if (!(hipSuccess == hipStreamCreate(sgemm_args[i].stream)))
        {
            CHECK_ERROR("hipStreamCreate");
        }
        sgemm_args[i].A_sz = A_sz;
        sgemm_args[i].B_sz = B_sz;
        sgemm_args[i].C_sz = C_sz;
        sgemm_args[i].matArow = matArow;
        sgemm_args[i].matAcol = matAcol;
        sgemm_args[i].matBrow = matBrow;
        sgemm_args[i].matBcol = matBcol;
        sgemm_args[i].matA = matA;
        sgemm_args[i].matBT = matBT;

        if (!(hipSuccess == hipStreamCreate(mriq_args[i].stream)))
        {
            CHECK_ERROR("hipStreamCreate");
        }

        mriq_args[i].numX = numX;
        mriq_args[i].numK = numK;
        mriq_args[i].kx = kx;
        mriq_args[i].ky = ky;
        mriq_args[i].kz = kz;
        mriq_args[i].x = x;
        mriq_args[i].y = y;
        mriq_args[i].z = z;
        mriq_args[i].phiR = phiR;
        mriq_args[i].phiI = phiI;
        mriq_args[i].phiMag = phiMag;
        mriq_args[i].Qr = Qr;
        mriq_args[i].Qi = Qi;
    }

    hipEventRecord(start_event, 0);
    for (int i = 0; i < num_threads / 2; ++i)
    {
        if (pthread_create(&threads[2 * i], NULL, launch_kernel_sgemm, &sgemm_args[i]))
        {
            fprintf(stderr, "Error creating threadn");
            return 1;
        }

        if (pthread_create(&threads[2 * i + 1], NULL, launch_kernel_mriq, &mriq_args[i]))
        {
            fprintf(stderr, "Error creating threadn");
            return 1;
        }
    }

    for (int i = 0; i < num_threads; ++i)
    {
        if (pthread_join(threads[i], NULL))
        {
            fprintf(stderr, "Error joining threadn");
            return 2;
        }
    }

    if (!(hipSuccess == hipEventRecord(stop_event, 0)))
    {
        CHECK_ERROR("hipEventRecord");
    }

    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);

    for (int i = 0; i < num_threads / 2; ++i)
    {
        hipStreamDestroy(*(sgemm_args[i].stream));
        hipStreamDestroy(*(mriq_args[i].stream));
    }

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipDeviceReset();

    printf("Measured time for sample = %.3fms\n", elapsed_time);

    // free(phiMag);
    // free(kx);
    // free(ky);
    // free(kz);
    // free(x);
    // free(y);
    // free(z);
    // free(phiR);
    // free(phiI);
    // free(Qr);
    // free(Qi);

    return 0;
}
