#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/*
 * Kernel of dense matrix-matrix multiplication kernel.
 * The algorithm is based on CUDA sgemm code from Vasily Volkov
 * at UC Berkeley.
 */

#define CHECK_ERROR(errorMessage)                                               \
    {                                                                           \
        hipError_t err = hipGetLastError();                                   \
        if (hipSuccess != err)                                                 \
        {                                                                       \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",   \
                    errorMessage, __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    }

// CML x RML = CML, baseline version, 510FLOP/s on Fermi
/* Pseudo code
for i < M ; i += 64   // thread block.x
 for j < N; j += 16   // thread block.y
  for tx = 0; tx < 16; tx++ // thread index x; tile of M loop
  for ty = 0; ty < 4 ; ty++ // thread index y; tile of M loop

  for m < 16; m += 1;
     c[m] = 0.0f

  for k < K; k += 4   // seq

   b[ty][tx] = B[k+ty][j+tx]

   for l < 4; l +=1   // seq
    for m < 16; m +=1 // seq
      c[m] += A[i+ty*16+tx][k+l]+b[l][m]

*/

// Parameters of tile sizes
#define TILE_N 16
#define TILE_TB_HEIGHT 8
#define TILE_M (TILE_N * TILE_TB_HEIGHT)

__global__ void mysgemmNT(const float *A, int lda, const float *B, int ldb, float *C, int ldc, int k, float alpha, float beta, dim3 blockOffset)
{
    // Partial results
    float c[TILE_N];
    for (int i = 0; i < TILE_N; i++)
        c[i] = 0.0f;
    int mid = threadIdx.y * blockDim.x + threadIdx.x; // flattened id
    int m = (blockOffset.x + blockIdx.x) * TILE_M + mid;
    int n = (blockOffset.y + blockIdx.y) * TILE_N + threadIdx.x;
    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];
    for (int i = 0; i < k; i += TILE_TB_HEIGHT)
    {
        float a;
        b_s[threadIdx.y][threadIdx.x] = B[n + (i + threadIdx.y) * ldb];
        __syncthreads();
        for (int j = 0; j < TILE_TB_HEIGHT; j++)
        {
            a = A[m + (i + j) * lda];
            for (int kk = 0; kk < TILE_N; kk++)
                c[kk] += a * b_s[j][kk];
        }
        __syncthreads();
    }
    int t = ldc * (blockOffset.y + blockIdx.y) * TILE_N + m;
    for (int i = 0; i < TILE_N; i++)
    {
        C[t + i * ldc] = C[t + i * ldc] * beta + alpha * c[i];
    }
}

void regtileSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc, hipStream_t *stream)
{
    if ((transa != 'N') && (transa != 'n'))
    {
        std::cerr << "unsupported value of 'transa' in regtileSgemm()" << std::endl;
        return;
    }

    if ((transb != 'T') && (transb != 't'))
    {
        std::cerr << "unsupported value of 'transb' in regtileSgemm()" << std::endl;
        return;
    }

    int m_slicer = 2, n_slicer = 6;
    // In this code we assume the matrix sizes are multiple of tile size
    if ((m % (TILE_M * m_slicer)) || (n % (TILE_N * n_slicer)))
    {
        std::cerr << "unsupported size of matrix. m should be multiple of " << TILE_M * m_slicer
                  << "; n should be multiple of " << TILE_N * n_slicer << std::endl;
    }

    dim3 gridConf(m / TILE_M, n / TILE_N);
    dim3 blockConf(TILE_N, TILE_TB_HEIGHT);
    dim3 sGridConf(m / (TILE_M * m_slicer), n / (TILE_N * n_slicer));

    printf("gridConf: (%d, %d)\n", gridConf.x, gridConf.y);
    printf("blockConf: (%d, %d)\n", blockConf.x, blockConf.y);
    printf("sGridConf: (%d, %d)\n", sGridConf.x, sGridConf.y);

    dim3 blockOffset(0, 0);
    while (blockOffset.x < m / TILE_M && blockOffset.y < n / TILE_N)
    {
        if (stream == nullptr)
            mysgemmNT<<<sGridConf, blockConf>>>(A, lda, B, ldb, C, ldc, k, alpha, beta, blockOffset);
        else
            mysgemmNT<<<sGridConf, blockConf, 0, *stream>>>(A, lda, B, ldb, C, ldc, k, alpha, beta, blockOffset);

        blockOffset.x += sGridConf.x;
        while (blockOffset.x >= gridConf.x)
        {
            blockOffset.x -= gridConf.x;
            blockOffset.y += sGridConf.y;
        }
    }

    CHECK_ERROR("mySgemm");
}
