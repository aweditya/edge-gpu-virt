#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <malloc.h>
#include <vector>
#include <parboil.h>
#include "sgemm_kernel_sliced.cu"
#include "computeQ_sliced.cu"
#include "file.h"

// I/O routines
extern bool readColMajorMatrixFile(const char *fn, int &nr_row, int &nr_col, std::vector<float> &v);
extern bool writeColMajorMatrixFile(const char *fn, int, int, std::vector<float> &);

extern "C" void computeGold(float *, const float *, const float *, unsigned int, unsigned int, unsigned int);

typedef struct thread_args_sgemm
{
    size_t A_sz, B_sz, C_sz;
    int matArow, matAcol;
    int matBrow, matBcol;
    std::vector<float> matA, matBT;
} thread_args_sgemm_t;

typedef struct thread_args_mriq
{
    int numX, numK;      /* Number of X and K values */
    float *kx, *ky, *kz; /* K trajectory (3D vectors) */
    float *x, *y, *z;    /* X coordinates (3D vectors) */
    float *phiR, *phiI;  /* Phi values (complex) */
    float *phiMag;       /* Magnitude of Phi */
    float *Qr, *Qi;      /* Q signal (complex) */

} thread_args_mriq_t;

#define CHECK_ERROR(errorMessage)                                               \
    {                                                                           \
        hipError_t err = hipGetLastError();                                   \
        if (hipSuccess != err)                                                 \
        {                                                                       \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",   \
                    errorMessage, __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    }

void *launch_kernel_sgemm(void *thread_args)
{
    float *dA, *dB, *dC;
    thread_args_sgemm_t *args = (thread_args_sgemm_t *)thread_args;

    // CUDA memory allocation
    std::vector<float> matC(args->matArow * args->matBcol);

    if (!(hipSuccess == hipMalloc((void **)&dA, args->A_sz)))
    {
        CHECK_ERROR("hipMalloc");
    }

    if (!(hipSuccess == hipMalloc((void **)&dB, args->B_sz)))
    {
        CHECK_ERROR("hipMalloc");
    }

    if (!(hipSuccess == hipMalloc((void **)&dC, args->C_sz)))
    {
        CHECK_ERROR("hipMalloc");
    }

    // Copy A and B^T into device memory
    if (!(hipSuccess == hipMemcpy(dA, &(args->matA.front()), args->A_sz, hipMemcpyHostToDevice)))
    {
        CHECK_ERROR("hipMemcpy");
    }

    if (!(hipSuccess == hipMemcpy(dB, &(args->matBT.front()), args->B_sz, hipMemcpyHostToDevice)))
    {
        CHECK_ERROR("hipMemcpy");
    }

    // Use standard sgemm interface
    regtileSgemm('N', 'T', args->matArow, args->matBcol, args->matAcol, 1.0f,
                 dA, args->matArow, dB, args->matBcol, 0.0f, dC, args->matArow, nullptr);

    if (!(hipSuccess == hipMemcpy(&matC.front(), dC, args->C_sz, hipMemcpyDeviceToHost)))
    {
        CHECK_ERROR("hipMemcpy");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return NULL;
}

void *launch_kernel_mriq(void *thread_args)
{
    thread_args_mriq_t *args = (thread_args_mriq_t *)thread_args;

    /* Create CPU data structures */
    createDataStructsCPU(args->numK, args->numX, &(args->phiMag), &(args->Qr), &(args->Qi));

    /* GPU section 1 (precompute PhiMag) */
    {
        /* Mirror several data structures on the device */
        float *phiR_d, *phiI_d;
        float *phiMag_d;

        if (!(hipSuccess == hipMalloc((void **)&phiR_d, args->numK * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        if (!(hipSuccess == hipMalloc((void **)&phiI_d, args->numK * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        if (!(hipSuccess == hipMemcpy(phiR_d, args->phiR, args->numK * sizeof(float), hipMemcpyHostToDevice)))
        {
            CHECK_ERROR("hipMemcpy");
        }

        if (!(hipSuccess == hipMemcpy(phiI_d, args->phiI, args->numK * sizeof(float), hipMemcpyHostToDevice)))
        {
            CHECK_ERROR("hipMemcpy");
        }

        if (!(hipSuccess == hipMalloc((void **)&phiMag_d, args->numK * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        hipDeviceSynchronize();

        computePhiMag_GPU(args->numK, phiR_d, phiI_d, phiMag_d, nullptr);

        hipDeviceSynchronize();

        if (!(hipSuccess == hipMemcpy(args->phiMag, phiMag_d, args->numK * sizeof(float), hipMemcpyDeviceToHost)))
        {
            CHECK_ERROR("hipMemcpy");
        }

        hipFree(phiMag_d);
        hipFree(phiR_d);
        hipFree(phiI_d);
    }

    struct kValues *kVals;
    kVals = (struct kValues *)calloc(args->numK, sizeof(struct kValues));
    for (int k = 0; k < args->numK; k++)
    {
        kVals[k].Kx = args->kx[k];
        kVals[k].Ky = args->ky[k];
        kVals[k].Kz = args->kz[k];
        kVals[k].PhiMag = args->phiMag[k];
    }

    /* GPU section 2 */
    {
        float *x_d, *y_d, *z_d;
        float *Qr_d, *Qi_d;

        if (!(hipSuccess == hipMalloc((void **)&x_d, args->numX * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        if (!(hipSuccess == hipMemcpy(x_d, args->x, args->numX * sizeof(float), hipMemcpyHostToDevice)))
        {
            CHECK_ERROR("hipMemcpy");
        }

        if (!(hipSuccess == hipMalloc((void **)&y_d, args->numX * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        if (!(hipSuccess == hipMemcpy(y_d, args->y, args->numX * sizeof(float), hipMemcpyHostToDevice)))
        {
            CHECK_ERROR("hipMemcpy");
        }

        if (!(hipSuccess == hipMalloc((void **)&z_d, args->numX * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        if (!(hipSuccess == hipMemcpy(z_d, args->z, args->numX * sizeof(float), hipMemcpyHostToDevice)))
        {
            CHECK_ERROR("hipMemcpy");
        }

        if (!(hipSuccess == hipMalloc((void **)&Qr_d, args->numX * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        if (!(hipSuccess == hipMemset((void *)Qr_d, 0, args->numX * sizeof(float))))
        {
            CHECK_ERROR("hipMemset");
        }

        if (!(hipSuccess == hipMalloc((void **)&Qi_d, args->numX * sizeof(float))))
        {
            CHECK_ERROR("hipMalloc");
        }

        if (!(hipSuccess == hipMemset((void *)Qi_d, 0, args->numX * sizeof(float))))
        {
            CHECK_ERROR("hipMemset");
        }

        hipDeviceSynchronize();

        computeQ_GPU(args->numK, args->numX, x_d, y_d, z_d, kVals, Qr_d, Qi_d, nullptr);

        hipDeviceSynchronize();

        if (!(hipSuccess == hipMemcpy(args->Qr, Qr_d, args->numX * sizeof(float), hipMemcpyDeviceToHost)))
        {
            CHECK_ERROR("hipMemcpy");
        }

        if (!(hipSuccess == hipMemcpy(args->Qi, Qi_d, args->numX * sizeof(float), hipMemcpyDeviceToHost)))
        {
            CHECK_ERROR("hipMemcpy");
        }

        hipFree(x_d);
        hipFree(y_d);
        hipFree(z_d);
        hipFree(Qr_d);
        hipFree(Qi_d);
    }

    free(kVals);
    return NULL;
}

int main(int argc, char *argv[])
{
    struct pb_Parameters *params;

    size_t A_sz, B_sz, C_sz;
    int matArow, matAcol;
    int matBrow, matBcol;
    std::vector<float> matA, matBT;

    int numX, numK;      /* Number of X and K values */
    int original_numK;   /* Number of K values in input file */
    float *kx, *ky, *kz; /* K trajectory (3D vectors) */
    float *x, *y, *z;    /* X coordinates (3D vectors) */
    float *phiR, *phiI;  /* Phi values (complex) */
    float *phiMag;       /* Magnitude of Phi */
    float *Qr, *Qi;      /* Q signal (complex) */

    /* Read command line. Expect 3 inputs: A, B and B^T
       in column-major layout*/
    params = pb_ReadParameters(&argc, argv);
    printf("%s %s %s\n", params->inpFiles[0], params->inpFiles[1], params->inpFiles[2]);
    if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] == NULL) || (params->inpFiles[2] == NULL) || (params->inpFiles[3] != NULL))
    {
        fprintf(stderr, "Expecting three input filenames\n");
        exit(-1);
    }

    inputData(params->inpFiles[0],
              &original_numK, &numX,
              &kx, &ky, &kz,
              &x, &y, &z,
              &phiR, &phiI);

    numK = original_numK;

    printf("%d pixels in output; %d samples in trajectory; using %d samples\n",
           numX, original_numK, numK);

    /* Read in data */
    // load A
    readColMajorMatrixFile(params->inpFiles[1],
                           matArow, matAcol, matA);
    // copy A to device memory
    A_sz = matArow * matAcol * sizeof(float);

    // load B^T
    readColMajorMatrixFile(params->inpFiles[2],
                           matBcol, matBrow, matBT);

    B_sz = matBrow * matBcol * sizeof(float);

    // allocate space for C
    C_sz = matArow * matBcol * sizeof(float);

    float elapsed_time;

    const int num_threads = 4;
    thread_args_sgemm_t sgemm_args[num_threads / 2];
    thread_args_mriq_t mriq_args[num_threads / 2];

    hipEvent_t start_event, stop_event;
    if (!(hipSuccess == hipEventCreate(&start_event)))
    {
        CHECK_ERROR("hipEventCreate");
    }

    if (!(hipSuccess == hipEventCreate(&stop_event)))
    {
        CHECK_ERROR("hipEventCreate");
    }

    for (int i = 0; i < num_threads / 2; ++i)
    {
        sgemm_args[i].A_sz = A_sz;
        sgemm_args[i].B_sz = B_sz;
        sgemm_args[i].C_sz = C_sz;
        sgemm_args[i].matArow = matArow;
        sgemm_args[i].matAcol = matAcol;
        sgemm_args[i].matBrow = matBrow;
        sgemm_args[i].matBcol = matBcol;
        sgemm_args[i].matA = matA;
        sgemm_args[i].matBT = matBT;

        mriq_args[i].numX = numX;
        mriq_args[i].numK = numK;
        mriq_args[i].kx = kx;
        mriq_args[i].ky = ky;
        mriq_args[i].kz = kz;
        mriq_args[i].x = x;
        mriq_args[i].y = y;
        mriq_args[i].z = z;
        mriq_args[i].phiR = phiR;
        mriq_args[i].phiI = phiI;
        mriq_args[i].phiMag = phiMag;
        mriq_args[i].Qr = Qr;
        mriq_args[i].Qi = Qi;
    }

    hipEventRecord(start_event, 0);
    for (int i = 0; i < num_threads / 2; ++i)
    {
        launch_kernel_sgemm(&sgemm_args[i]);
        launch_kernel_mriq(&mriq_args[i]);
    }

    if (!(hipSuccess == hipEventRecord(stop_event, 0)))
    {
        CHECK_ERROR("hipEventRecord");
    }

    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipDeviceReset();

    printf("Measured time for sample = %.3fms\n", elapsed_time);

    // free(phiMag);
    // free(kx);
    // free(ky);
    // free(kz);
    // free(x);
    // free(y);
    // free(z);
    // free(phiR);
    // free(phiI);
    // free(Qr);
    // free(Qi);

    return 0;
}
