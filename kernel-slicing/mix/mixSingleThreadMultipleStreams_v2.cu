#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <malloc.h>
#include <vector>
#include <parboil.h>
#include "sgemm_kernel_sliced.cu"
#include "computeQ_sliced.cu"
#include "file.h"

// I/O routines
extern bool readColMajorMatrixFile(const char *fn, int &nr_row, int &nr_col, std::vector<float> &v);
extern bool writeColMajorMatrixFile(const char *fn, int, int, std::vector<float> &);

extern "C" void computeGold(float *, const float *, const float *, unsigned int, unsigned int, unsigned int);

typedef struct thread_args_sgemm
{
    hipStream_t stream;
    size_t A_sz, B_sz, C_sz;
    int matArow, matAcol;
    int matBrow, matBcol;
    std::vector<float> matA, matBT;
} thread_args_sgemm_t;

typedef struct thread_args_mriq
{
    hipStream_t stream;
    int numX, numK;      /* Number of X and K values */
    float *kx, *ky, *kz; /* K trajectory (3D vectors) */
    float *x, *y, *z;    /* X coordinates (3D vectors) */
    float *phiR, *phiI;  /* Phi values (complex) */
    float *phiMag;       /* Magnitude of Phi */
    float *Qr, *Qi;      /* Q signal (complex) */

} thread_args_mriq_t;

#define CHECK_ERROR(errorMessage)                                               \
    {                                                                           \
        hipError_t err = hipGetLastError();                                   \
        if (hipSuccess != err)                                                 \
        {                                                                       \
            fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",   \
                    errorMessage, __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    }

struct timeval t0, t1, t2;

float *dA, *dB, *dC;
void *launch_kernel_sgemm(void *thread_args)
{
    thread_args_sgemm_t *args = (thread_args_sgemm_t *)thread_args;

    // CUDA memory allocation
    std::vector<float> matC(args->matArow * args->matBcol);

    // Copy A and B^T into device memory
    if (!(hipSuccess == hipMemcpyAsync(dA, &(args->matA.front()), args->A_sz, hipMemcpyHostToDevice, args->stream)))
    {
        CHECK_ERROR("hipMemcpyAsync");
    }

    if (!(hipSuccess == hipMemcpyAsync(dB, &(args->matBT.front()), args->B_sz, hipMemcpyHostToDevice, args->stream)))
    {
        CHECK_ERROR("hipMemcpyAsync");
    }

    // Use standard sgemm interface
    regtileSgemm('N', 'T', args->matArow, args->matBcol, args->matAcol, 1.0f,
                 dA, args->matArow, dB, args->matBcol, 0.0f, dC, args->matArow, &args->stream);

    if (!(hipSuccess == hipMemcpyAsync(&matC.front(), dC, args->C_sz, hipMemcpyDeviceToHost, args->stream)))
    {
        CHECK_ERROR("hipMemcpyAsync");
    }
    
    gettimeofday(&t1, NULL);
    return NULL;
}

float *phiR_d, *phiI_d;
float *phiMag_d;

float *x_d, *y_d, *z_d;
float *Qr_d, *Qi_d;

void *launch_kernel_mriq(void *thread_args)
{
    thread_args_mriq_t *args = (thread_args_mriq_t *)thread_args;

    /* GPU section 1 (precompute PhiMag) */
    {
        /* Mirror several data structures on the device */
        if (!(hipSuccess == hipMemcpyAsync(phiR_d, args->phiR, args->numK * sizeof(float), hipMemcpyHostToDevice, args->stream)))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        if (!(hipSuccess == hipMemcpyAsync(phiI_d, args->phiI, args->numK * sizeof(float), hipMemcpyHostToDevice, args->stream)))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        computePhiMag_GPU(args->numK, phiR_d, phiI_d, phiMag_d, &args->stream);

        if (!(hipSuccess == hipMemcpyAsync(args->phiMag, phiMag_d, args->numK * sizeof(float), hipMemcpyDeviceToHost, args->stream)))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }
    }

    struct kValues *kVals;
    kVals = (struct kValues *)calloc(args->numK, sizeof(struct kValues));
    for (int k = 0; k < args->numK; k++)
    {
        kVals[k].Kx = args->kx[k];
        kVals[k].Ky = args->ky[k];
        kVals[k].Kz = args->kz[k];
        kVals[k].PhiMag = args->phiMag[k];
    }

    /* GPU section 2 */
    {
        if (!(hipSuccess == hipMemcpyAsync(x_d, args->x, args->numX * sizeof(float), hipMemcpyHostToDevice, args->stream)))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        if (!(hipSuccess == hipMemcpyAsync(y_d, args->y, args->numX * sizeof(float), hipMemcpyHostToDevice, args->stream)))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        if (!(hipSuccess == hipMemcpyAsync(z_d, args->z, args->numX * sizeof(float), hipMemcpyHostToDevice, args->stream)))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        if (!(hipSuccess == hipMemsetAsync((void *)Qr_d, 0, args->numX * sizeof(float), args->stream)))
        {
            CHECK_ERROR("hipMemsetAsync");
        }

        if (!(hipSuccess == hipMemsetAsync((void *)Qi_d, 0, args->numX * sizeof(float), args->stream)))
        {
            CHECK_ERROR("hipMemsetAsync");
        }

        computeQ_GPU(args->numK, args->numX, x_d, y_d, z_d, kVals, Qr_d, Qi_d, &args->stream);

        if (!(hipSuccess == hipMemcpyAsync(args->Qr, Qr_d, args->numX * sizeof(float), hipMemcpyDeviceToHost, args->stream)))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }

        if (!(hipSuccess == hipMemcpyAsync(args->Qi, Qi_d, args->numX * sizeof(float), hipMemcpyDeviceToHost, args->stream)))
        {
            CHECK_ERROR("hipMemcpyAsync");
        }
    }

    free(kVals);
    gettimeofday(&t2, NULL);
    return NULL;
}

int main(int argc, char *argv[])
{
    struct pb_Parameters *params;

    size_t A_sz, B_sz, C_sz;
    int matArow, matAcol;
    int matBrow, matBcol;
    std::vector<float> matA, matBT;

    int numX, numK;      /* Number of X and K values */
    int original_numK;   /* Number of K values in input file */
    float *kx, *ky, *kz; /* K trajectory (3D vectors) */
    float *x, *y, *z;    /* X coordinates (3D vectors) */
    float *phiR, *phiI;  /* Phi values (complex) */
    float *phiMag;       /* Magnitude of Phi */
    float *Qr, *Qi;      /* Q signal (complex) */

    /* Read command line. Expect 3 inputs: A, B and B^T
       in column-major layout*/
    params = pb_ReadParameters(&argc, argv);
    printf("%s %s %s\n", params->inpFiles[0], params->inpFiles[1], params->inpFiles[2]);
    if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] == NULL) || (params->inpFiles[2] == NULL) || (params->inpFiles[3] != NULL))
    {
        fprintf(stderr, "Expecting three input filenames\n");
        exit(-1);
    }

    inputData(params->inpFiles[0],
              &original_numK, &numX,
              &kx, &ky, &kz,
              &x, &y, &z,
              &phiR, &phiI);

    numK = original_numK;

    printf("%d pixels in output; %d samples in trajectory; using %d samples\n",
           numX, original_numK, numK);

    /* Read in data */
    // load A
    readColMajorMatrixFile(params->inpFiles[1],
                           matArow, matAcol, matA);
    // copy A to device memory
    A_sz = matArow * matAcol * sizeof(float);

    // load B^T
    readColMajorMatrixFile(params->inpFiles[2],
                           matBcol, matBrow, matBT);

    B_sz = matBrow * matBcol * sizeof(float);

    // allocate space for C
    C_sz = matArow * matBcol * sizeof(float);

    /* Memory allocation for SGEMM */
    if (!(hipSuccess == hipMalloc((void **)&dA, A_sz)))
    {
        CHECK_ERROR("hipMalloc");
    }

    if (!(hipSuccess == hipMalloc((void **)&dB, B_sz)))
    {
        CHECK_ERROR("hipMalloc");
    }

    if (!(hipSuccess == hipMalloc((void **)&dC, C_sz)))
    {
        CHECK_ERROR("hipMalloc");
    }
    /*******************************/

    /* Memory allocation for MRI-Q */

    /* Create CPU data structures */
    createDataStructsCPU(numK, numX, &phiMag, &Qr, &Qi);

    if (!(hipSuccess == hipMalloc((void **)&phiR_d, numK * sizeof(float))))
    {
        CHECK_ERROR("hipMalloc");
    }

    if (!(hipSuccess == hipMalloc((void **)&phiI_d, numK * sizeof(float))))
    {
        CHECK_ERROR("hipMalloc");
    }

    if (!(hipSuccess == hipMalloc((void **)&phiMag_d, numK * sizeof(float))))
    {
        CHECK_ERROR("hipMalloc");
    }

    if (!(hipSuccess == hipMalloc((void **)&x_d, numX * sizeof(float))))
    {
        CHECK_ERROR("hipMalloc");
    }

    if (!(hipSuccess == hipMalloc((void **)&y_d, numX * sizeof(float))))
    {
        CHECK_ERROR("hipMalloc");
    }

    if (!(hipSuccess == hipMalloc((void **)&z_d, numX * sizeof(float))))
    {
        CHECK_ERROR("hipMalloc");
    }

    if (!(hipSuccess == hipMalloc((void **)&Qr_d, numX * sizeof(float))))
    {
        CHECK_ERROR("hipMalloc");
    }
    if (!(hipSuccess == hipMalloc((void **)&Qi_d, numX * sizeof(float))))
    {
        CHECK_ERROR("hipMalloc");
    }
    /******************************/

    float elapsed_time;

    hipEvent_t start_event, stop_event;
    if (!(hipSuccess == hipEventCreate(&start_event)))
    {
        CHECK_ERROR("hipEventCreate");
    }

    thread_args_sgemm_t sgemm_args;
    thread_args_mriq_t mriq_args;
    if (!(hipSuccess == hipEventCreate(&stop_event)))
    {
        CHECK_ERROR("hipEventCreate");
    }

    if (!(hipSuccess == hipStreamCreate(&sgemm_args.stream)))
    {
        CHECK_ERROR("hipStreamCreate");
    }
    sgemm_args.A_sz = A_sz;
    sgemm_args.B_sz = B_sz;
    sgemm_args.C_sz = C_sz;
    sgemm_args.matArow = matArow;
    sgemm_args.matAcol = matAcol;
    sgemm_args.matBrow = matBrow;
    sgemm_args.matBcol = matBcol;
    sgemm_args.matA = matA;
    sgemm_args.matBT = matBT;

    if (!(hipSuccess == hipStreamCreate(&mriq_args.stream)))
    {
        CHECK_ERROR("hipStreamCreate");
    }

    mriq_args.numX = numX;
    mriq_args.numK = numK;
    mriq_args.kx = kx;
    mriq_args.ky = ky;
    mriq_args.kz = kz;
    mriq_args.x = x;
    mriq_args.y = y;
    mriq_args.z = z;
    mriq_args.phiR = phiR;
    mriq_args.phiI = phiI;
    mriq_args.phiMag = phiMag;
    mriq_args.Qr = Qr;
    mriq_args.Qi = Qi;

    hipEventRecord(start_event, 0);

    gettimeofday(&t0, NULL);
    launch_kernel_sgemm(&sgemm_args);
    launch_kernel_mriq(&mriq_args);

    if (!(hipSuccess == hipEventRecord(stop_event, 0)))
    {
        CHECK_ERROR("hipEventRecord");
    }

    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);

    hipStreamDestroy(sgemm_args.stream);
    hipStreamDestroy(mriq_args.stream);

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    /* Free SGEMM memory */
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    /*********************/

    /* Free MRI-Q memory */
    hipFree(phiMag_d);
    hipFree(phiR_d);
    hipFree(phiI_d);

    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
    hipFree(Qr_d);
    hipFree(Qi_d);
    /*********************/

    hipDeviceReset();

    printf("Measured time for sample = %.3fms\n", elapsed_time);
    
    struct timeval dt01, dt02;
    timersub(&t1, &t0, &dt01);
    timersub(&t2, &t0, &dt02);
    printf("Time taken for completion of SGEMM: %ld.%06ld\n", dt01.tv_sec, dt01.tv_usec);
    printf("Time taken for completion of MRI-Q: %ld.%06ld\n", dt02.tv_sec, dt02.tv_usec);

    free(phiMag);
    free(kx);
    free(ky);
    free(kz);
    free(x);
    free(y);
    free(z);
    free(phiR);
    free(phiI);
    free(Qr);
    free(Qi);

    return 0;
}
