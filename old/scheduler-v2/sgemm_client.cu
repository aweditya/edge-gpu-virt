#include "sgemm_client.h"
#include "errchk.h"

#include <vector>

void SGEMMClient::setupKernel()
{
    // CUDA memory allocation
    checkCudaErrors(hipMalloc((void **)&dA, A_sz));
    checkCudaErrors(hipMalloc((void **)&dB, B_sz));
    checkCudaErrors(hipMalloc((void **)&dC, C_sz));

    // Copy A and B^T into device memory
    checkCudaErrors(hipMemcpyAsync(dA, &matA.front(), A_sz, hipMemcpyHostToDevice, *(kernel.clientStream)));
    checkCudaErrors(hipMemcpyAsync(dB, &matBT.front(), B_sz, hipMemcpyHostToDevice, *(kernel.clientStream)));
}

void SGEMMClient::finishKernel()
{
    // Copy C into host memory
    checkCudaErrors(hipMemcpyAsync(&(matC.front()), dC, C_sz, hipMemcpyDeviceToHost, *(kernel.clientStream)));

    // CUDA memory deallocation
    checkCudaErrors(hipFree(dA));
    checkCudaErrors(hipFree(dB));
    checkCudaErrors(hipFree(dC));
}