#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include "sgemm_client.h"

/*
 * Kernel of dense matrix-matrix multiplication kernel.
 * The algorithm is based on CUDA sgemm code from Vasily Volkov
 * at UC Berkeley.
 */

// CML x RML = CML, baseline version, 510FLOP/s on Fermi
/* Pseudo code
for i < M ; i += 64   // thread block.x
 for j < N; j += 16   // thread block.y
  for tx = 0; tx < 16; tx++ // thread index x; tile of M loop
  for ty = 0; ty < 4 ; ty++ // thread index y; tile of M loop

  for m < 16; m += 1;
     c[m] = 0.0f

  for k < K; k += 4   // seq

   b[ty][tx] = B[k+ty][j+tx]

   for l < 4; l +=1   // seq
    for m < 16; m +=1 // seq
      c[m] += A[i+ty*16+tx][k+l]+b[l][m]

*/

__global__ void mysgemmNT(const float *A, int lda, const float *B, int ldb, float *C, int ldc, int k, float alpha, float beta, int blockOffsetx, int blockOffsety)
{
    // Partial results
    float c[TILE_N];
    for (int i = 0; i < TILE_N; i++)
        c[i] = 0.0f;
    int mid = threadIdx.y * blockDim.x + threadIdx.x; // flattened id
    int m = (blockOffsetx + blockIdx.x) * TILE_M + mid;
    int n = (blockOffsety + blockIdx.y) * TILE_N + threadIdx.x;
    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];
    for (int i = 0; i < k; i += TILE_TB_HEIGHT)
    {
        float a;
        b_s[threadIdx.y][threadIdx.x] = B[n + (i + threadIdx.y) * ldb];
        __syncthreads();
        for (int j = 0; j < TILE_TB_HEIGHT; j++)
        {
            a = A[m + (i + j) * lda];
            for (int kk = 0; kk < TILE_N; kk++)
                c[kk] += a * b_s[j][kk];
        }
        __syncthreads();
    }
    int t = ldc * (blockOffsety + blockIdx.y) * TILE_N + m;
    for (int i = 0; i < TILE_N; i++)
    {
        C[t + i * ldc] = C[t + i * ldc] * beta + alpha * c[i];
    }
}
