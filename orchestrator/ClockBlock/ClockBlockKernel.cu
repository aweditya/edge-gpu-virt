#include "ClockBlockKernel.h"

void ClockBlockKernel::memAlloc()
{
    h_a = (long *)malloc(sizeof(long));
    checkCudaErrors(hipMalloc(&d_a, sizeof(long)));

    clock_count = KERNEL_TIME * clockRate;

    args[3] = &blockDimX;
    args[4] = &blockDimY;
    args[5] = &blockDimZ;
    args[6] = perSMThreads;
    args[7] = &d_a;
    args[8] = &clock_count;
}

void ClockBlockKernel::memcpyHtoD(const hipStream_t &stream)
{
}

void ClockBlockKernel::memcpyDtoH(const hipStream_t &stream)
{
    checkCudaErrors(hipMemcpyDtoHAsync(h_a, d_a, sizeof(long), stream));
}

void ClockBlockKernel::memFree()
{
    checkCudaErrors(hipFree(d_a));

    free(h_a);
}