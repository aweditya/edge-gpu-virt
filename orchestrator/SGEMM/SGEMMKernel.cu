#include "SGEMMKernel.h"

void SGEMMKernel::memAlloc()
{
    matC = std::vector<float>(C_sz);

    checkCudaErrors(hipMalloc(&dA, A_sz));
    checkCudaErrors(hipMalloc(&dB, B_sz));
    checkCudaErrors(hipMalloc(&dC, C_sz));

    args[3] = &blockDimX;
    args[4] = &blockDimY;
    args[5] = &blockDimZ;
    args[6] = perSMThreads;
    args[7] = &dA;
    args[8] = &matArow;
    args[9] = &dB;
    args[10] = &matBcol;
    args[11] = &dC;
    args[12] = &matArow;
    args[13] = &matAcol;
    args[14] = &alpha;
    args[15] = &beta;
}

void SGEMMKernel::memcpyHtoD(const hipStream_t &stream)
{
    checkCudaErrors(hipMemcpyHtoDAsync(dA, &(matA.front()), A_sz, stream));
    checkCudaErrors(hipMemcpyHtoDAsync(dB, &(matBT.front()), B_sz, stream));
}

void SGEMMKernel::memcpyDtoH(const hipStream_t &stream)
{
    checkCudaErrors(hipMemcpyDtoHAsync(&(matC.front()), dC, C_sz, stream));
}

void SGEMMKernel::memFree()
{
    checkCudaErrors(hipFree(dA));
    checkCudaErrors(hipFree(dB));
    checkCudaErrors(hipFree(dC));
}