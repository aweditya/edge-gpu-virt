#include "SGEMMKernel.h"

void SGEMMKernel::memAlloc()
{
    matC = std::vector<float>(C_sz);

    checkCudaErrors(hipMalloc(&dA, A_sz));
    checkCudaErrors(hipMalloc(&dB, B_sz));
    checkCudaErrors(hipMalloc(&dC, C_sz));

    args[3] = &dA;
    args[4] = &matArow;
    args[5] = &dB;
    args[6] = &matBcol;
    args[7] = &dC;
    args[8] = &matArow;
    args[9] = &matAcol;
    args[10] = &alpha;
    args[11] = &beta;
}

void SGEMMKernel::memcpyHtoD(const hipStream_t &stream)
{
    checkCudaErrors(hipMemcpyHtoDAsync(dA, &(matA.front()), A_sz, stream));
    checkCudaErrors(hipMemcpyHtoDAsync(dB, &(matBT.front()), B_sz, stream));
}

void SGEMMKernel::memcpyDtoH(const hipStream_t &stream)
{
    checkCudaErrors(hipMemcpyDtoHAsync(&(matC.front()), dC, C_sz, stream));
}

void SGEMMKernel::memFree()
{
    checkCudaErrors(hipFree(dA));
    checkCudaErrors(hipFree(dB));
    checkCudaErrors(hipFree(dC));
}