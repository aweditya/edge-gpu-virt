#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include "errchk.h"

#define NUM_KERNELS 2                                // Number of kernels
#define KERNEL_TIME 1000L                            // Time to run kernel for (in ms)
#define LOGGING_INTERVAL 10                          // Logging interval (in ms)
#define LOGGING_DURATION (NUM_KERNELS * KERNEL_TIME) // Logging duration (in ms)

typedef struct kernel_thread_args
{
    int kernelID;
    long clock_count;
    hipStream_t stream;
} kernel_thread_args_t;

__device__ int d_threadsRunning[NUM_KERNELS];

// This is a kernel that does no real work but runs at least for a specified number of clocks
__global__ void clockBlock(long clock_count, int kernelID)
{
    atomicAdd(&d_threadsRunning[kernelID], 1); // Increment when thread starts
    printf("[%d] %d\n", kernelID, d_threadsRunning[kernelID]);

    unsigned int start_clock = (unsigned int)clock();

    long clock_offset = 0;
    while (clock_offset < clock_count)
    {
        unsigned int end_clock = (unsigned int)clock();

        // The code below should work like
        // this (thanks to modular arithmetics):
        //
        // clock_offset = (clock_t) (end_clock > start_clock ?
        //                           end_clock - start_clock :
        //                           end_clock + (0xffffffffu - start_clock));
        //
        // Indeed, let m = 2^32 then
        // end - start = end + m - start (mod m).

        clock_offset = (long)(end_clock - start_clock);
    }

    atomicSub(&d_threadsRunning[kernelID], 1); // Decrement when thread starts
    printf("[%d] %d\n", kernelID, d_threadsRunning[kernelID]);
}

// Periodically report number of threads for each kernel
__host__ void reportThreadsRunning()
{
    int h_threadsRunning[NUM_KERNELS];
    checkCudaErrors(hipMemcpyFromSymbolAsync(h_threadsRunning, HIP_SYMBOL(d_threadsRunning), sizeof(int) * NUM_KERNELS));
    // for (int i = 0; i < NUM_KERNELS; ++i)
    // {
    //     printf("%d\t", h_threadsRunning[i]);
    // }
    // printf("\n");
}

void *kernel_threadfunction(void *args)
{
    kernel_thread_args_t *kernel_args = (kernel_thread_args_t *)args;
    printf("[%ld] launched kernel #%d\n", (unsigned long)pthread_self(), kernel_args->kernelID);
    clockBlock<<<1, 32, 0, kernel_args->stream>>>(kernel_args->clock_count, kernel_args->kernelID);
    return NULL;
}

void *logger_threadfunction(void *args)
{
    const int iterations = LOGGING_DURATION / LOGGING_INTERVAL;
    for (int i = 0; i < iterations; ++i)
    {
        reportThreadsRunning();
        usleep(LOGGING_INTERVAL * 1000);
    }

    return NULL;
}

int main()
{
    hipDeviceProp_t prop;
    checkCudaErrors(hipGetDeviceProperties(&prop, 0));
    int clockRate = prop.clockRate;
    int clock_count = KERNEL_TIME * clockRate;

    // Initialise d_threadsRunning
    int *_d_threadsRunning;
    checkCudaErrors(hipGetSymbolAddress((void **)&_d_threadsRunning, d_threadsRunning));
    checkCudaErrors(hipMemset(_d_threadsRunning, 0, sizeof(int) * NUM_KERNELS));

    kernel_thread_args_t args[NUM_KERNELS];
    pthread_t kernel_threads[NUM_KERNELS], logger_thread;

    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        args[i].kernelID = i;
        args[i].clock_count = clock_count;
        checkCudaErrors(hipStreamCreate(&args[i].stream));
    }

    /* Start logger thread */
    if (pthread_create(&logger_thread, NULL, logger_threadfunction, NULL) != 0)
    {
        perror("pthread_create failed");
    }

    /* Launch kernels */
    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        if (pthread_create(&kernel_threads[i], NULL, kernel_threadfunction, &args[i]) != 0)
        {
            perror("pthread_create failed");
        }
    }

    if (pthread_join(logger_thread, NULL) != 0)
    {
        perror("pthread_join failed");
    }

    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        if (pthread_join(kernel_threads[i], NULL) != 0)
        {
            perror("pthread_join failed");
        }
    }

    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        checkCudaErrors(hipStreamDestroy(args[i].stream));
    }

    checkCudaErrors(hipDeviceReset());

    return 0;
}
