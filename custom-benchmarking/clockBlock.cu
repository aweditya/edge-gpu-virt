#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include "errchk.h"

#define NUM_KERNELS 16                               // Number of kernels
#define KERNEL_TIME 1000L                            // Time to run kernel for (in ms)
#define LOGGING_INTERVAL 10                          // Logging interval (in ms)
#define LOGGING_DURATION (NUM_KERNELS * KERNEL_TIME) // Logging duration (in ms)

typedef struct kernel_thread_args
{
    int *threadsRunning;
    long clock_count;
    hipStream_t stream;
} kernel_thread_args_t;

// This is a kernel that does no real work but runs at least for a specified number of clocks
__global__ void clockBlock(long clock_count, int *threadsRunning)
{
    atomicAdd(threadsRunning, 1); // Increment when thread starts

    unsigned int start_clock = (unsigned int)clock();

    long clock_offset = 0;
    while (clock_offset < clock_count)
    {
        unsigned int end_clock = (unsigned int)clock();

        // The code below should work like
        // this (thanks to modular arithmetics):
        //
        // clock_offset = (clock_t) (end_clock > start_clock ?
        //                           end_clock - start_clock :
        //                           end_clock + (0xffffffffu - start_clock));
        //
        // Indeed, let m = 2^32 then
        // end - start = end + m - start (mod m).

        clock_offset = (long)(end_clock - start_clock);
    }

    atomicSub(threadsRunning, 1); // Decrement when thread starts
}

// Periodically report number of threads for each kernel
__host__ void reportThreadsRunning(int *allThreadsRunning)
{
    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        printf("%d\t", allThreadsRunning[i]);
    }
    printf("\n");
}

void *kernel_threadfunction(void *args)
{
    kernel_thread_args_t *kernel_args = (kernel_thread_args_t *)args;
    clockBlock<<<32, 128, 0, kernel_args->stream>>>(kernel_args->clock_count, kernel_args->threadsRunning);
    return NULL;
}

void *logger_threadfunction(void *args)
{
    int *allThreadsRunning = (int *)args;
    const int iterations = LOGGING_DURATION / LOGGING_INTERVAL;
    for (int i = 0; i < iterations; ++i)
    {
        reportThreadsRunning(allThreadsRunning);
        usleep(LOGGING_INTERVAL * 1000);
    }

    return NULL;
}

int main()
{
    hipDeviceProp_t prop;
    checkCudaErrors(hipGetDeviceProperties(&prop, 0));
    int clockRate = prop.clockRate;
    int clock_count = KERNEL_TIME * clockRate;

    // Allocate unified memory
    int *allThreadsRunning = NULL;
    checkCudaErrors(hipMallocManaged((void **)&allThreadsRunning, sizeof(int) * NUM_KERNELS));

    kernel_thread_args_t args[NUM_KERNELS];
    pthread_t kernel_threads[NUM_KERNELS], logger_thread;

    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        args[i].threadsRunning = &allThreadsRunning[i];
        args[i].clock_count = clock_count;
        checkCudaErrors(hipStreamCreate(&args[i].stream));
    }

    /* Start logger thread */
    if (pthread_create(&logger_thread, NULL, logger_threadfunction, allThreadsRunning) != 0)
    {
        perror("pthread_create failed");
    }

    /* Launch kernels */
    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        if (pthread_create(&kernel_threads[i], NULL, kernel_threadfunction, &args[i]) != 0)
        {
            perror("pthread_create failed");
        }
    }

    if (pthread_join(logger_thread, NULL) != 0)
    {
        perror("pthread_join failed");
    }

    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        if (pthread_join(kernel_threads[i], NULL) != 0)
        {
            perror("pthread_join failed");
        }
    }

    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        checkCudaErrors(hipStreamDestroy(args[i].stream));
    }

    checkCudaErrors(hipFree(allThreadsRunning));
    checkCudaErrors(hipDeviceReset());

    return 0;
}
