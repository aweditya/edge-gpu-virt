#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include "errchk.h"

#define NUM_KERNELS 2                                // Number of kernels
#define KERNEL_TIME 1000L                            // Time to run kernel for (in ms)
#define LOGGING_INTERVAL 10                          // Logging interval (in ms)
#define LOGGING_DURATION (NUM_KERNELS * KERNEL_TIME) // Logging duration (in ms)

int multiProcessorCount;

typedef struct kernel_thread_args
{
    long clock_count;
    hipStream_t stream;
    int *perSMThreads;
} kernel_thread_args;

// Get ID of SM on which kernel thread is running
__device__ unsigned int get_smid(void)
{
    unsigned int smID;
    asm volatile("mov.u32 %0, %smid;" : "=r"(smID));
    return smID;
}

// This is a kernel that does no real work but runs at least for a specified number of clocks
__global__ void clockBlock(long clock_count, int *perSMThreads)
{
    int smID;
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) // Call only once for each thread block
    {
        smID = get_smid();
        atomicAdd(&perSMThreads[smID], 1); // Increment number of blocks running on current SM
    }

    unsigned int start_clock = (unsigned int)clock();

    long clock_offset = 0;
    while (clock_offset < clock_count)
    {
        unsigned int end_clock = (unsigned int)clock();

        // The code below should work like
        // this (thanks to modular arithmetics):
        //
        // clock_offset = (clock_t) (end_clock > start_clock ?
        //                           end_clock - start_clock :
        //                           end_clock + (0xffffffffu - start_clock));
        //
        // Indeed, let m = 2^32 then
        // end - start = end + m - start (mod m).

        clock_offset = (long)(end_clock - start_clock);
    }

    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
    {
        atomicSub(&perSMThreads[smID], 1); // Decrement number of blocks running on current SM
    }
}
void *kernel_threadfunction(void *args)
{
    kernel_thread_args *kernel_args = (kernel_thread_args *)args;
    clockBlock<<<32, 128, 0, kernel_args->stream>>>(kernel_args->clock_count, kernel_args->perSMThreads);
    return NULL;
}

// Periodically report number of threads for each kernel
__host__ void reportThreadsRunning(int *perSMThreads)
{
    for (int i = 0; i < multiProcessorCount; ++i)
    {
        printf("%d\t", perSMThreads[i]);
    }
    printf("\n");
}

void *logger_threadfunction(void *args)
{
    int *perSMThreads = (int *)args;
    const int iterations = LOGGING_DURATION / LOGGING_INTERVAL;
    for (int i = 0; i < iterations; ++i)
    {
        reportThreadsRunning(perSMThreads);
        usleep(LOGGING_INTERVAL * 1000);
    }

    return NULL;
}

int main()
{
    hipDeviceProp_t prop;
    checkCudaErrors(hipGetDeviceProperties(&prop, 0));

    multiProcessorCount = prop.multiProcessorCount;
    int clockRate = prop.clockRate;
    int clock_count = KERNEL_TIME * clockRate;

    // Allocate unified memory
    int *perSMThreads = NULL;
    checkCudaErrors(hipMallocManaged((void **)&perSMThreads, sizeof(int) * multiProcessorCount));

    kernel_thread_args args[NUM_KERNELS];
    pthread_t kernel_threads[NUM_KERNELS], logger_thread;

    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        args[i].clock_count = clock_count;
        checkCudaErrors(hipStreamCreate(&args[i].stream));

        args[i].perSMThreads = perSMThreads;
    }

    /* Start logger thread */
    if (pthread_create(&logger_thread, NULL, logger_threadfunction, perSMThreads) != 0)
    {
        perror("pthread_create failed");
    }

    /* Launch kernels */
    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        if (pthread_create(&kernel_threads[i], NULL, kernel_threadfunction, &args[i]) != 0)
        {
            perror("pthread_create failed");
        }
    }

    if (pthread_join(logger_thread, NULL) != 0)
    {
        perror("pthread_join failed");
    }

    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        if (pthread_join(kernel_threads[i], NULL) != 0)
        {
            perror("pthread_join failed");
        }
    }

    for (int i = 0; i < NUM_KERNELS; ++i)
    {
        checkCudaErrors(hipStreamDestroy(args[i].stream));
    }

    checkCudaErrors(hipFree(perSMThreads));
    checkCudaErrors(hipDeviceReset());

    return 0;
}
