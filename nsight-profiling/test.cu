#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <pthread.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define DEVICE_RESET hipDeviceReset();

template <typename T>
void check(T result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        fprintf(stderr, "CUDA error at %s:%d code=%d \"%s\" \n",
                file, line, static_cast<unsigned int>(result), func);
        DEVICE_RESET
        // Make sure we call CUDA Device Reset before exiting
        exit(EXIT_FAILURE);
    }
}

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

typedef struct
{
    hipStream_t stream;
    int offset;
} targs_t;

__global__ void test_kernel(int *a, int offset, int N)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < N)
    {
        a[tid] += offset;
    }
}

void *thread_func(void *thread_args)
{
    struct timeval t0, t1, dt;
    gettimeofday(&t0, NULL);

    targs_t *targs = (targs_t *)(thread_args);
    
    int N = 4 * 1024 * 1024;

    int *d_a = 0;
    checkCudaErrors(hipMalloc((void **)&d_a, N * sizeof(int)));
    checkCudaErrors(hipMemset(d_a, 0, N * sizeof(int)));
    test_kernel<<<4 * 1024, 1024, 0, targs->stream>>>(d_a, targs->offset, N);

    int *a = (int *)malloc(N * sizeof(int));
    checkCudaErrors(hipMemcpyAsync(a, d_a, N * sizeof(int), hipMemcpyDeviceToHost, targs->stream));

    hipFree(d_a);
    free(a);

    gettimeofday(&t1, NULL);
    timersub(&t1, &t0, &dt);
    printf("thread_func (thread %ld) took %ld.%06ld sec\n", (long)pthread_self(), dt.tv_sec, dt.tv_usec);

    return NULL;
}

int main()
{
    float elapsed = 0;

    pthread_t threads[2];
    targs_t targs[2];

    for (int i = 0; i < 2; ++i)
    {
        hipStreamCreate(&(targs[i].stream));
        targs[i].offset = i + 1;
    }

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipEventRecord(start, 0));
    for (int i = 0; i < 2; ++i)
    {
        if (pthread_create(&(threads[i]), NULL, thread_func, &(targs[i])))
        {
            fprintf(stderr, "Error creating threadn");
            return 1;
        }
    }

    for (int i = 0; i < 2; ++i)
    {
        if (pthread_join(threads[i], NULL))
        {
            fprintf(stderr, "Error joining threadn");
            return 2;
        }
    }

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsed, start, stop));

    for (int i = 0; i < 2; ++i)
    {
        hipStreamDestroy(targs[i].stream);
    }

    printf("measured time for sample = %.3fms\n", elapsed);
    DEVICE_RESET

    return 0;
}
