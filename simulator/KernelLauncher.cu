#include "KernelLauncher.h"

void *KernelLauncher::threadFunction()
{
    checkCudaErrors(hipModuleLoad(&module, moduleFile.c_str()));
    checkCudaErrors(hipModuleGetFunction(&function, module, kernelName.c_str()));
    callback->memAlloc();
    callback->memcpyHtoD(stream);
    launchKernel();
    callback->memcpyDtoH(stream);
    callback->memFree();

    return NULL;
}