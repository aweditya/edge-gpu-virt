#include "hip/hip_runtime.h"
#include "matrixAdd.h"

__global__ void matrxAdd(double *a, double *b, double *c, int blockOffset)
{
    int tid = (blockIdx.x + blockOffset) * blockDim.x + threadIdx.x;
    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}