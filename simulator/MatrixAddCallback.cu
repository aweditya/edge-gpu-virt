#include "MatrixAddCallback.h"

void MatrixAddCallback::memAlloc()
{
    h_a = (double *)malloc(N * sizeof(double));
    h_b = (double *)malloc(N * sizeof(double));
    h_c = (double *)malloc(N * sizeof(double));

    checkCudaErrors(hipMalloc(&d_a, N * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_b, N * sizeof(double)));
    checkCudaErrors(hipMalloc(&d_c, N * sizeof(double)));
}

void MatrixAddCallback::memcpyHtoD()
{
    checkCudaErrors(hipMemcpyHtoDAsync(d_a, h_a, N * sizeof(double), stream));
    checkCudaErrors(hipMemcpyHtoDAsync(d_b, h_b, N * sizeof(double), stream));
}

void MatrixAddCallback::memcpyDtoH()
{
    checkCudaErrors(hipMemcpyDtoHAsync(h_c, d_c, N * sizeof(double), stream));
}

void MatrixAddCallback::memFree()
{
    checkCudaErrors(hipFree(d_a));
    checkCudaErrors(hipFree(d_b));
    checkCudaErrors(hipFree(d_c));

    free(h_a);
    free(h_b);
    free(h_c);
}