#include "hip/hip_runtime.h"
#include "Scheduler.h"

void Scheduler::scheduleKernel(kernel_attr_t *kernel)
{
    kernel->id = rand();
    set_state(&(kernel->kcb), LAUNCH);

    pthread_mutex_lock(&mutex);
    activeKernels.push_back(kernel);
    pthread_mutex_unlock(&mutex);
}

void Scheduler::launchKernel(kernel_attr_t *kernel)
{
    for (int i = 0; i < min(kernel->kcb.slicesToLaunch, kernel->kcb.totalSlices); ++i)
    {
        printf("[kernel id: %d] slices left = %d\n", kernel->id, kernel->kcb.totalSlices);
        checkCudaErrors(hipModuleLaunchKernel(kernel->function,
                                       kernel->sGridDimX,
                                       kernel->sGridDimY,
                                       kernel->sGridDimZ,
                                       kernel->blockDimX,
                                       kernel->blockDimY,
                                       kernel->blockDimZ,
                                       kernel->sharedMemBytes,
                                       kernel->stream,
                                       kernel->kernelParams,
                                       nullptr));

        kernel->blockOffsetX += kernel->sGridDimX;
        while (kernel->blockOffsetX >= kernel->gridDimX)
        {
            kernel->blockOffsetX -= kernel->gridDimX;
            kernel->blockOffsetY += kernel->sGridDimY;
        }

        while (kernel->blockOffsetY >= kernel->gridDimY)
        {
            kernel->blockOffsetY -= kernel->gridDimY;
            kernel->blockOffsetZ += kernel->sGridDimZ;
        }
    }

    kernel->kcb.totalSlices = max(kernel->kcb.totalSlices - kernel->kcb.slicesToLaunch, 0);
}