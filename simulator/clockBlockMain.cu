#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include "KernelWrapper.h"
#include "ClockBlockKernel.h"
#include "FCFSScheduler.h"

hipDevice_t device;
int clockRate;
hipCtx_t context;
size_t totalGlobalMem;

void initCuda()
{
    int deviceCount = 0;
    checkCudaErrors(hipInit(0));
    int major = 0, minor = 0;

    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stderr, "Error: no devices supporting CUDA\n");
        exit(-1);
    }

    // get first CUDA device
    checkCudaErrors(hipDeviceGet(&device, 0));
    char name[100];
    hipDeviceGetName(name, 100, device);
    printf("> Using device 0: %s\n", name);

    // get device properties
    checkCudaErrors(hipDeviceGetAttribute(&clockRate, hipDeviceAttributeClockRate, device));

    // get compute capabilities and the devicename
    checkCudaErrors(hipDeviceComputeCapability(&major, &minor, device));
    printf("> GPU Device has SM %d.%d compute capability\n", major, minor);

    checkCudaErrors(hipDeviceTotalMem(&totalGlobalMem, device));
    printf("  Total amount of global memory:   %llu bytes\n",
           (unsigned long long)totalGlobalMem);
    printf("  64-bit Memory Address:           %s\n",
           (totalGlobalMem > (unsigned long long)4 * 1024 * 1024 * 1024L) ? "YES" : "NO");

    checkCudaErrors(hipCtxCreate(&context, 0, device));
}

void finishCuda()
{
    cuCtxDetach(context);
}

int main(int argc, char **argv)
{
    initCuda();
    srand(0);

    FCFSScheduler scheduler;

    const std::string moduleFile1 = "./ptx/clockBlock1.ptx";
    const std::string moduleFile2 = "./ptx/clockBlock2.ptx";
    const std::string kernelName = "clockBlock";

    hipStream_t stream1, stream2;
    checkCudaErrors(hipStreamCreateWithFlags(&stream1, hipStreamDefault));
    checkCudaErrors(hipStreamCreateWithFlags(&stream2, hipStreamDefault));

    ClockBlockKernel clockBlockKernel1(clockRate), clockBlockKernel2(clockRate);
    kernel_attr_t attr1 = {
        .gridDimX = 8,
        .gridDimY = 1,
        .gridDimZ = 1,
        .blockDimX = 128,
        .blockDimY = 1,
        .blockDimZ = 1,
        .sGridDimX = 8 / 4,
        .sGridDimY = 1,
        .sGridDimZ = 1,
        .sharedMemBytes = 0,
        .stream = stream1};

    kernel_attr_t attr2 = {
        .gridDimX = 8,
        .gridDimY = 1,
        .gridDimZ = 1,
        .blockDimX = 128,
        .blockDimY = 1,
        .blockDimZ = 1,
        .sGridDimX = 8 / 4,
        .sGridDimY = 1,
        .sGridDimZ = 1,
        .sharedMemBytes = 0,
        .stream = stream2};

    KernelWrapper wrapper1(&scheduler, context, moduleFile1, kernelName, &attr1, &clockBlockKernel1);
    KernelWrapper wrapper2(&scheduler, context, moduleFile2, kernelName, &attr2, &clockBlockKernel2);

    scheduler.run();
    wrapper1.launch();
    wrapper2.launch();

    wrapper1.finish();
    wrapper2.finish();

    scheduler.stop();
    scheduler.finish();

    checkCudaErrors(hipStreamDestroy(stream1));
    checkCudaErrors(hipStreamDestroy(stream2));
    finishCuda();

    return 0;
}
