#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include "KernelWrapper.h"
#include "MatrixAddKernel.h"

hipDevice_t device;
hipCtx_t context;
size_t totalGlobalMem;

void initCuda()
{
    int deviceCount = 0;
    checkCudaErrors(hipInit(0));
    int major = 0, minor = 0;

    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stderr, "Error: no devices supporting CUDA\n");
        exit(-1);
    }

    // get first CUDA device
    checkCudaErrors(hipDeviceGet(&device, 0));
    char name[100];
    hipDeviceGetName(name, 100, device);
    printf("> Using device 0: %s\n", name);

    // get compute capabilities and the devicename
    checkCudaErrors(hipDeviceComputeCapability(&major, &minor, device));
    printf("> GPU Device has SM %d.%d compute capability\n", major, minor);

    checkCudaErrors(hipDeviceTotalMem(&totalGlobalMem, device));
    printf("  Total amount of global memory:   %llu bytes\n",
           (unsigned long long)totalGlobalMem);
    printf("  64-bit Memory Address:           %s\n",
           (totalGlobalMem > (unsigned long long)4 * 1024 * 1024 * 1024L) ? "YES" : "NO");

    checkCudaErrors(hipCtxCreate(&context, 0, device));
}

void finishCuda()
{
    cuCtxDetach(context);
}

int main(int argc, char **argv)
{
    initCuda();
    srand(0);

    bool done = false;
    Scheduler scheduler(&done);

    const std::string moduleFile1 = "./ptx/matrixAdd1.ptx";
    const std::string moduleFile2 = "./ptx/matrixAdd2.ptx";
    const std::string kernelName = "matrixAdd";

    hipStream_t stream1, stream2;
    checkCudaErrors(hipStreamCreateWithFlags(&stream1, hipStreamDefault));
    checkCudaErrors(hipStreamCreateWithFlags(&stream2, hipStreamDefault));

    MatrixAddKernel matrixAddKernel1, matrixAddKernel2;
    kernel_attr_t attr1 = {
        .gridDimX = N,
        .gridDimY = 1,
        .gridDimZ = 1,
        .blockDimX = 1,
        .blockDimY = 1,
        .blockDimZ = 1,
        .sGridDimX = N / 16,
        .sGridDimY = 1,
        .sGridDimZ = 1,
        .sharedMemBytes = 0,
        .stream = stream1};

    kernel_attr_t attr2 = {
        .gridDimX = N,
        .gridDimY = 1,
        .gridDimZ = 1,
        .blockDimX = 1,
        .blockDimY = 1,
        .blockDimZ = 1,
        .sGridDimX = N / 16,
        .sGridDimY = 1,
        .sGridDimZ = 1,
        .sharedMemBytes = 0,
        .stream = stream2};

    KernelWrapper wrapper1(&scheduler, context, moduleFile1, kernelName, &attr1, &matrixAddKernel1);
    KernelWrapper wrapper2(&scheduler, context, moduleFile2, kernelName, &attr2, &matrixAddKernel2);

    scheduler.run();
    wrapper1.launch();
    wrapper2.launch();

    wrapper1.finish();
    wrapper2.finish();
    
    done = true;
    scheduler.finish();

    checkCudaErrors(hipStreamDestroy(stream1));
    checkCudaErrors(hipStreamDestroy(stream2));
    finishCuda();

    return 0;
}
