#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include "KernelWrapper.h"
#include "MatrixAddKernel.h"

hipDevice_t device;
hipCtx_t context;
size_t totalGlobalMem;

void initCuda()
{
    int deviceCount = 0;
    checkCudaErrors(hipInit(0));
    int major = 0, minor = 0;

    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stderr, "Error: no devices supporting CUDA\n");
        exit(-1);
    }

    // get first CUDA device
    checkCudaErrors(hipDeviceGet(&device, 0));
    char name[100];
    hipDeviceGetName(name, 100, device);
    printf("> Using device 0: %s\n", name);

    // get compute capabilities and the devicename
    checkCudaErrors(hipDeviceComputeCapability(&major, &minor, device));
    printf("> GPU Device has SM %d.%d compute capability\n", major, minor);

    checkCudaErrors(hipDeviceTotalMem(&totalGlobalMem, device));
    printf("  Total amount of global memory:   %llu bytes\n",
           (unsigned long long)totalGlobalMem);
    printf("  64-bit Memory Address:           %s\n",
           (totalGlobalMem > (unsigned long long)4 * 1024 * 1024 * 1024L) ? "YES" : "NO");

    checkCudaErrors(hipCtxCreate(&context, 0, device));
}

void finishCuda()
{
    cuCtxDetach(context);
}

int main(int argc, char **argv)
{
    const std::string moduleFile1 = "./ptx/matrixAdd1.ptx";
    const std::string kernelName = "matrixAdd";

    srand(0);
    Scheduler scheduler;

    initCuda();

    hipStream_t stream1;
    checkCudaErrors(hipStreamCreateWithFlags(&stream1, hipStreamDefault));

    MatrixAddKernel matrixAddKernel1;
    kernel_attr_t attr1 = {
        .gridDimX = N,
        .gridDimY = 1,
        .gridDimZ = 1,
        .blockDimX = 1,
        .blockDimY = 1,
        .blockDimZ = 1,
        .sGridDimX = N / 16,
        .sGridDimY = 1,
        .sGridDimZ = 1,
        .sharedMemBytes = 0,
        .stream = stream1};

    KernelWrapper wrapper1(&scheduler, context, moduleFile1, kernelName, &attr1, &matrixAddKernel1);

    wrapper1.launch();
    while (true)
    {
        if (scheduler.activeKernels.size() == 0)
        {
            continue;
        }
        else
        {        
            attr1.kcb.slicesToLaunch = 2;
            scheduler.launchKernel(&attr1);

            if (attr1.kcb.totalSlices == 0)
            {
                pthread_mutex_lock(&(attr1.kcb.kernel_lock));
                attr1.kcb.state = MEMCPYDTOH;
                pthread_cond_signal(&(attr1.kcb.kernel_signal));
                pthread_mutex_unlock(&(attr1.kcb.kernel_lock));

                break;
            }
        }
    }

    wrapper1.finish();

    checkCudaErrors(hipStreamDestroy(stream1));
    finishCuda();

    return 0;
}
