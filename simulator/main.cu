#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include "KernelWrapper.h"
#include "MatrixAddKernel.h"

hipDevice_t device;
hipCtx_t context;
size_t totalGlobalMem;

void initCuda()
{
    int deviceCount = 0;
    checkCudaErrors(hipInit(0));
    int major = 0, minor = 0;

    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stderr, "Error: no devices supporting CUDA\n");
        exit(-1);
    }

    // get first CUDA device
    checkCudaErrors(hipDeviceGet(&device, 0));
    char name[100];
    hipDeviceGetName(name, 100, device);
    printf("> Using device 0: %s\n", name);

    // get compute capabilities and the devicename
    checkCudaErrors(hipDeviceComputeCapability(&major, &minor, device));
    printf("> GPU Device has SM %d.%d compute capability\n", major, minor);

    checkCudaErrors(hipDeviceTotalMem(&totalGlobalMem, device));
    printf("  Total amount of global memory:   %llu bytes\n",
           (unsigned long long)totalGlobalMem);
    printf("  64-bit Memory Address:           %s\n",
           (totalGlobalMem > (unsigned long long)4 * 1024 * 1024 * 1024L) ? "YES" : "NO");

    checkCudaErrors(hipCtxCreate(&context, 0, device));
}

void finishCuda()
{
    cuCtxDetach(context);
}

int main(int argc, char **argv)
{
    const std::string moduleFile1 = "./ptx/matrixAdd1.ptx";
    const std::string moduleFile2 = "./ptx/matrixAdd2.ptx";
    const std::string kernelName = "matrixAdd";

    srand(0);
    Scheduler scheduler;

    initCuda();

    hipStream_t stream1, stream2;
    checkCudaErrors(hipStreamCreateWithFlags(&stream1, hipStreamDefault));
    checkCudaErrors(hipStreamCreateWithFlags(&stream2, hipStreamDefault));

    MatrixAddKernel matrixAddKernel1, matrixAddKernel2;
    kernel_attr_t attr1 = {
        .gridDimX = N,
        .gridDimY = 1,
        .gridDimZ = 1,
        .blockDimX = 1,
        .blockDimY = 1,
        .blockDimZ = 1,
        .sGridDimX = N / 16,
        .sGridDimY = 1,
        .sGridDimZ = 1,
        .sharedMemBytes = 0,
        .stream = stream1};

    kernel_attr_t attr2 = {
        .gridDimX = N,
        .gridDimY = 1,
        .gridDimZ = 1,
        .blockDimX = 1,
        .blockDimY = 1,
        .blockDimZ = 1,
        .sGridDimX = N / 16,
        .sGridDimY = 1,
        .sGridDimZ = 1,
        .sharedMemBytes = 0,
        .stream = stream2};



    int time = 0;
    KernelWrapper wrapper1(&scheduler, context, moduleFile1, kernelName, &attr1, &matrixAddKernel1);
    KernelWrapper wrapper2(&scheduler, context, moduleFile2, kernelName, &attr2, &matrixAddKernel2);

    wrapper1.launch();
    wrapper2.launch();
    while (true)
    {
        if (scheduler.activeKernels.size() == 0)
        {
            continue;
        }
        else
        {
            printf("[thread id: %ld] number of kernels: %ld\n", pthread_self(), scheduler.activeKernels.size());
            if (scheduler.activeKernels.size() == 1)
            {
                scheduler.activeKernels[0]->kcb.slicesToLaunch = 2;
                scheduler.launchKernel(scheduler.activeKernels[0]);

                if (scheduler.activeKernels[0]->kcb.totalSlices == 0)
                {
                    pthread_mutex_lock(&(scheduler.activeKernels[0]->kcb.kernel_lock));
                    scheduler.activeKernels[0]->kcb.state = MEMCPYDTOH;
                    pthread_cond_signal(&(scheduler.activeKernels[0]->kcb.kernel_signal));
                    pthread_mutex_unlock(&(scheduler.activeKernels[0]->kcb.kernel_lock));
                    scheduler.activeKernels.erase(scheduler.activeKernels.begin());
                }
            }
            else
            {
                scheduler.activeKernels[time % 2]->kcb.slicesToLaunch = (time % 2 + 1) * 2;
                scheduler.launchKernel(scheduler.activeKernels[time % 2]);

                if (scheduler.activeKernels[time % 2]->kcb.totalSlices == 0)
                {
                    pthread_mutex_lock(&(scheduler.activeKernels[time % 2]->kcb.kernel_lock));
                    scheduler.activeKernels[time % 2]->kcb.state = MEMCPYDTOH;
                    pthread_cond_signal(&(scheduler.activeKernels[time % 2]->kcb.kernel_signal));
                    pthread_mutex_unlock(&(scheduler.activeKernels[time % 2]->kcb.kernel_lock));
                    scheduler.activeKernels.erase(scheduler.activeKernels.begin() + time % 2);
                }
            }

            if (scheduler.activeKernels.size() == 0)
            {
                break;
            }

            time++;
        }
    }

    wrapper1.finish();
    wrapper2.finish();

    checkCudaErrors(hipStreamDestroy(stream1));
    checkCudaErrors(hipStreamDestroy(stream2));
    finishCuda();

    return 0;
}
