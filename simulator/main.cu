#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include "KernelLauncher.h"
#include "MatrixAddCallback.h"

hipDevice_t device;
hipCtx_t context;
size_t totalGlobalMem;

void initCuda()
{
    int deviceCount = 0;
    checkCudaErrors(hipInit(0));
    int major = 0, minor = 0;

    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stderr, "Error: no devices supporting CUDA\n");
        exit(-1);
    }

    // get first CUDA device
    checkCudaErrors(hipDeviceGet(&device, 0));
    char name[100];
    hipDeviceGetName(name, 100, device);
    printf("> Using device 0: %s\n", name);

    // get compute capabilities and the devicename
    checkCudaErrors(hipDeviceComputeCapability(&major, &minor, device));
    printf("> GPU Device has SM %d.%d compute capability\n", major, minor);

    checkCudaErrors(hipDeviceTotalMem(&totalGlobalMem, device));
    printf("  Total amount of global memory:   %llu bytes\n",
           (unsigned long long)totalGlobalMem);
    printf("  64-bit Memory Address:           %s\n",
           (totalGlobalMem > (unsigned long long)4 * 1024 * 1024 * 1024L) ? "YES" : "NO");

    checkCudaErrors(hipCtxCreate(&context, 0, device));
}

void finishCuda()
{
    cuCtxDetach(context);
}

int main(int argc, char **argv)
{
    srand(0);

    const std::string moduleFile1 = "./ptx/matrixAdd1.ptx";
    const std::string kernelName = "matrixAdd";

    initCuda();

    hipStream_t stream1;
    checkCudaErrors(hipStreamCreateWithFlags(&stream1, hipStreamDefault));

    MatrixAddCallback matrixAddCallback1;

    kernel_attr_t attr1 = {
        .gridDimX = N,
        .gridDimY = 1,
        .gridDimZ = 1,
        .blockDimX = 1,
        .blockDimY = 1,
        .blockDimZ = 1,
        .sGridDimX = N / 2,
        .sGridDimY = 1,
        .sGridDimZ = 1,
        .sharedMemBytes = 0,
        .stream = stream1};

    kernel_control_block_t kcb1;
    pthread_mutex_init(&(kcb1.kernel_lock), NULL);
    pthread_cond_init(&(kcb1.kernel_signal), NULL);

    KernelLauncher launcher1(rand(), &context, moduleFile1, kernelName, &attr1, &kcb1, &matrixAddCallback1);

    launcher1.launch();
    pthread_mutex_lock(&(launcher1.kcb->kernel_lock));
    while (launcher1.kcb->state == INIT)
    {
        pthread_cond_wait(&(launcher1.kcb->kernel_signal), &(launcher1.kcb->kernel_lock));
    }
    launcher1.kcb->state = LAUNCH;
    pthread_mutex_unlock(&(launcher1.kcb->kernel_lock));

    while (true)
    {
        launcher1.kcb->slicesToLaunch = 2;
        launcher1.launchKernel();

        if (launcher1.kcb->totalSlices == 0)
        {
            pthread_mutex_lock(&(launcher1.kcb->kernel_lock));
            launcher1.kcb->state = MEMCPYDTOH;
            pthread_cond_signal(&(launcher1.kcb->kernel_signal));
            pthread_mutex_unlock(&(launcher1.kcb->kernel_lock));
            break;
        }
    }

    launcher1.finish();

    pthread_mutex_destroy(&(launcher1.kcb->kernel_lock));
    pthread_cond_destroy(&(launcher1.kcb->kernel_signal));

    checkCudaErrors(hipStreamDestroy(stream1));
    finishCuda();

    return 0;
}
